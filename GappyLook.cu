#include "hip/hip_runtime.h"
#include "ComTypes.h"
#include "GappyLook.h"


__device__ int existPrecomputation(int* topPrecomputationList, int tokenA, int tokenB){
	int start = 0;
	int end = PRECOMPUTECOUNT-1;
	int middle;
	bool flag_a = true;
	while (end - start >= 0 && flag_a) {
		middle = (start + end) >> 1;
		if (topPrecomputationList[middle] > tokenA){
			end = middle - 1;						
		} else if (topPrecomputationList[middle] < tokenA){
			start = middle + 1;
		} else {
			flag_a = false;			
		}
	}

	start = 0;
	end = PRECOMPUTECOUNT-1;
	bool flag_b = true;
	int middle2;
	while (end - start >= 0 && flag_b) {
		middle2 = (start + end) >> 1;
		if (topPrecomputationList[middle2] > tokenB){
			end = middle2 - 1;						
		} else if (topPrecomputationList[middle2] < tokenB){
			start = middle2 + 1;
		} else {
			flag_b = false;			
		}
	}

	if(!flag_a && !flag_b){
		return middle*PRECOMPUTECOUNT+middle2;
	}
	return -1;
}


__device__ bool checkBoundaryGap(//Target checking required, not no interface changes.
		unsigned int start, 
		unsigned int ender,
		uint8_t* L_tar,
		uint8_t* R_tar,
		unsigned int* RLP){
	unsigned char L = 0;
	unsigned char R = 0;
	int sen_target_begin = -1;
	unsigned char min_L = 255;
	unsigned char max_R = 0;
	int tempind = 0;
	unsigned int temp;
	
	int target_start;
	int target_end;
	int k;
	bool returnVal;
	for(k = start; k <= ender; k++){
		temp = RLP[k];
		L = (temp >> 24) & 0xFF;
		R = (temp >> 16) & 0xFF;
		if ( (L == 255 || R == 255) && (k == start || k == ender) ){
			k = ender + 1;
			return false;
		} else if ( (L == 255 || R == 255) ) {
			L = 255;     
		} else if (k == start){		
			tempind = k - ((temp >> 8) & 0xFF) - 1 ;	
			//printf("L %u R %u - BK x %d y %d - bnum %d - tempind %d - k %d - Pi %u\n", L, R, blockIdx.x, blockIdx.y, bnum, tempind, k, ((RLP[k] >> 8) & 0xFF));
			if (tempind == -1){
				sen_target_begin = 0;
			} else {
				sen_target_begin = RLP[tempind];
			}
			min_L = L;
			max_R = R;
		} else {
			if (min_L > L) {
				min_L = L;
			}
			if (max_R < R) {
				max_R = R;
			}
		} 
	}

	if (min_L <= max_R && max_R - min_L < MAX_rule_span){
		tempind++;
		target_start = min_L + sen_target_begin;
		target_end = max_R + sen_target_begin;
		//printf("target start %d - tar end %d - sour start %d - sour end %d\n", ss, tt, current_str, t);
		//return consistent( (*target_start),  (*target_end), L_tar, R_tar, start, ender, tempind);
		//bool consistent(int start, int end, uint8_t* L_target, uint8_t* R_target, int start_chk, int end_chk, int startpos_source)
		returnVal = true;
		min_L = 255;
		max_R = 0;
		for(k = target_start; k <= target_end; k++){
			L = L_tar[k];
			R = R_tar[k];
			
			if (L==255 || R == 255){
				returnVal = true; //Change fixes original false to remove non-recog symbols.
			} else if (k == target_start){
				min_L = L;
				max_R = R;
			} else {
				if (min_L > L) {
					min_L = L;
				}
				if (max_R < R) {
					max_R = R;
				}
			} 
		}
			
		if(tempind+min_L != start || tempind+max_R != ender){
			returnVal = false;
		}
		return returnVal;
	}else {	
		return false;
	}
}

__global__ void oneGapLookUpSA(	
		int* refstr, 
		int* refsa,
		unsigned int toklen,
		int *qrysoffsettok,
		int qryscount,    
		int tokenscount,
		result_t_two* qryresult,
		int* connectoffset,
		int totalconnect,
		result_t* result_connect,
		int* tokindex_qryindex,
		oneGapOnSA* oneGapSA,	
		unsigned int precomp_onegap_count,
		int* frequentList,
		precomp_st_end* precomp_index,
		precompute_enu_3* precomp_onegap,
		unsigned int* counter,
		gappy_search* oneGapSearch,
		int distinctOneGapCount,
		unsigned int* RLP,
		uint8_t* L_tar,
		uint8_t* R_tar) {

	int blockId = blockIdx.y*gridDim.x+blockIdx.x;
	
	/*if(blockId == 0){
		for(int i = 0; i<distinctOneGapCount;i++){
			printf("BlockId %d - qrystart %d - searchInd %d - gap %d - startlen %d\n", 
				i, 
				oneGapSearch[i].qrystart, 
				oneGapSearch[i].qrystart+oneGapSearch[i].gap + oneGapSearch[i].qrystart_len,
				oneGapSearch[i].gap,
				oneGapSearch[i].qrystart_len);
		}
	}*/
	if (blockId >= distinctOneGapCount){
		return;
	}
	
	int longest_len_start = oneGapSearch[blockId].qrystart_len;
	int longest_len_end = oneGapSearch[blockId].qryend_len;
	int range1_up = -1;
	int range1_down = -1;
	int range2_up = -1;
	int range2_down = -1;

	int precomputationIndex = -1;
	int tokindex = oneGapSearch[blockId].qrystart;//aXb's a, first letter of a
	int search_tokindex = tokindex+oneGapSearch[blockId].gap + 
		oneGapSearch[blockId].qrystart_len;//aXb's b, first letter of b.
	int nextpos = -1;
	
	//if(threadIdx.x==0){ //threadIdx.x
	/*printf("BlockId %d - qrystart %d - searchInd %d - gap %d - startlen %d\n", 
			blockId, 
			tokindex, 
			search_tokindex,
			oneGapSearch[blockId].gap,
			oneGapSearch[blockId].qrystart_len);*/
	//}
	if(oneGapSearch[blockId].gap==0||tokindex <0/*||tokindex > 11||search_tokindex>11*/){
		printf("Wrong!!\n");
		/*printf("Wrong!!! BlockId %d - qrystart %d - searchInd %d - gap %d - startlen %d\n", 
			blockId, 
			tokindex, 
			search_tokindex,
			oneGapSearch[blockId].gap,
			oneGapSearch[blockId].qrystart_len);*/
		return;
	}
	int tobesearch_start;
	int tobesearch_end;
	int dis, dis2; 
	int threadx = threadIdx.x;
	int cc;
	bool forwardOrBack = true;

	//Sanity check
	if (qryresult[search_tokindex].longestmatch < longest_len_end || 
			qryresult[tokindex].longestmatch < longest_len_start){
		printf("Not possible in side one gap SA]\n");
		return;
	}

	//precomputation possible
	precomputationIndex = existPrecomputation(frequentList, 
			qrysoffsettok[qryscount+tokindex+longest_len_start-1], 
			qrysoffsettok[qryscount+search_tokindex]);		
	
	if (precomputationIndex == -1){
		if(longest_len_start == 1){
			range1_up = qryresult[tokindex].up;
			range1_down = qryresult[tokindex].down;
			dis = range1_down - range1_up;
		} else {
			cc=connectoffset[tokindex] + longest_len_start - 2;
			range1_up = result_connect[cc].up;
			range1_down = result_connect[cc].down;
			dis = range1_down - range1_up;
		}

		if(longest_len_end == 1){
			range2_up = qryresult[search_tokindex].up;
			range2_down = qryresult[search_tokindex].down;
			dis2 = range2_down - range2_up;
		} else {
			cc=connectoffset[search_tokindex] + longest_len_end - 2;
			range2_up = result_connect[cc].up;
			range2_down = result_connect[cc].down;
			dis2 = range2_down - range2_up;
		}

		if(dis <= dis2){
			tobesearch_start = range1_up;
			tobesearch_end = range1_down;
			forwardOrBack = true;
		} else {
			dis = dis2;
			tobesearch_start = range2_up;
			tobesearch_end = range2_down;
			forwardOrBack = false;
		}

	}else {
		tobesearch_start = precomp_index[precomputationIndex].start;
		tobesearch_end = precomp_index[precomputationIndex].end;
		dis = tobesearch_end - tobesearch_start;		
	}

	if(precomputationIndex != -1 && longest_len_start == 1 && longest_len_end == 1
		 && dis >= 0){
		//Sanity check
		if (precomputationIndex < 0){
			printf("Not possible! Precomp less than zero\n");
			return;
		}
		if (threadx == 0){
			nextpos = atomicAdd(counter, 1);
			oneGapSA[nextpos].position  = blockId;
			oneGapSA[nextpos].str_position = precomputationIndex;
			oneGapSA[nextpos].length = 0;
		}
		return;
	}	
	
	int move = 0;
	bool flager = true;
	int precompstart;
	int precomplen;

	int backoff = 0;
	bool stop = false;
	int forward = -1;
	int gostart = 0;
	int temp =-1;
	int matchcount;

	while(threadx <= dis){
		move = 0;
		flager = true;
		if (precomputationIndex != -1){
			precompstart = precomp_onegap[tobesearch_start+threadx].start;
			precomplen = precomp_onegap[tobesearch_start+threadx].length;

			if (precomplen + 1 + longest_len_start - 1 + 
					longest_len_end - 1 > MAX_rule_span ){
				flager = false;
			}

			//Check the previous tokens are the same otherwise stop
			if (flager && longest_len_start > 1){
				backoff = 0;
				stop = false;
				while(flager && !stop){
					backoff++;
					if(precompstart-backoff<0||refstr[precompstart-backoff]!=
							qrysoffsettok[qryscount+tokindex+longest_len_start-1-backoff]){
						flager= false;
					}
					if(longest_len_start - backoff <= 1){
						stop = true;
					}
				}
			}

			//Check the B and afterwards
			if(flager&& longest_len_end > 1){
				forward = 1;
				while(forward < longest_len_end && flager){
					forward++;
					if (refstr[precompstart + precomplen + forward-1] 
							!= qrysoffsettok[qryscount+search_tokindex+forward-1]){
						flager = false;
					}
				}
			}

			//Everything is OK, record results
			if(flager){
				nextpos = atomicAdd(counter, 1);
				oneGapSA[nextpos].position	= blockId;
				oneGapSA[nextpos].str_position = precompstart - 
						longest_len_start + 1;
				oneGapSA[nextpos].length = precomplen + longest_len_start - 1 + 
						longest_len_end - 1;  // to the end aXb's b					
			}
		} else if (forwardOrBack){
			///Forward search, start from aXb's a, because a's range is smaller.
			gostart = refsa[threadx+tobesearch_start];
			move = 0;
			temp = -1;
			while(flager){
				if(move == 0){
					//Check gap, if the gap is a <2 between two sentences, 
					//then skip
					temp = refstr[gostart+longest_len_start];
					if(temp < 2){
						flager = false;
					}
				}
				temp = refstr[gostart+longest_len_start+MIN_gap_size+move];
				if (temp < 2 ){
					flager = false;
				} else if (flager && temp == qrysoffsettok[qryscount+search_tokindex]){
					//onegap[nextpos].gap = longest_start_iter+MIN_gap_size+move;
					matchcount = 1;
					stop = false;
					while(!stop&&matchcount < longest_len_end
							&& longest_len_start+MIN_gap_size+move + 1 + matchcount <= MAX_rule_span){ 
						////debugging
						if (qrysoffsettok[qryscount+search_tokindex+matchcount]< 2){
							printf("This is not possible - onegap kernel");
							return;
						}
						//debugging
						
						backoff = refstr[gostart+longest_len_start+MIN_gap_size+move+matchcount];
						if (backoff < 2){
							stop = true;
							flager = false;
						} else if(backoff == 
								qrysoffsettok[qryscount+search_tokindex+matchcount]){
							matchcount++;
						}	else {
							stop = true;
						}				
					}

					if(matchcount == longest_len_end && 
						 checkBoundaryGap(gostart+longest_len_start, 
							gostart+longest_len_start+MIN_gap_size+move+longest_len_end-1-longest_len_end, 
							L_tar, 
							R_tar, 
							RLP)){
						nextpos = atomicAdd(counter, 1);
						oneGapSA[nextpos].position	= blockId;
						oneGapSA[nextpos].str_position = gostart;
						oneGapSA[nextpos].length = 
							longest_len_start+MIN_gap_size+move+longest_len_end-1; 
						// to the end aXb's b	
					}
				} 

				move++;//+move is the correct pos of b's starting pos
				if(longest_len_start+MIN_gap_size+move + longest_len_end > MAX_rule_span){
					flager = false;
				}
			}						
		} else {
			///Backward search, start from aXb's b's first character, because b's range is smaller.
			gostart = refsa[threadx+tobesearch_start];
			move = 0;
			temp = -1;
			while(flager){
				if(move == 0){
					//Check gap, if the gap is a <2 between two sentences, 
					//then skip
					temp = refstr[gostart-1];
					if(temp < 2){
						flager = false;
					}
				}
				if(gostart-1-MIN_gap_size-move<0){
					temp = -1;
				} else {
					temp = refstr[gostart-1-MIN_gap_size-move];
				}
				
				if (temp < 2 ){
					flager = false;
				} else if (flager && temp == qrysoffsettok[qryscount+tokindex+longest_len_start-1]){
					//onegap[nextpos].gap = longest_start_iter+MIN_gap_size+move;
					matchcount = 1;
					stop = false;
					while(!stop&&matchcount < longest_len_start
							&& longest_len_end+MIN_gap_size+move + 1 + matchcount <= MAX_rule_span){
						///debuginig
						if (qrysoffsettok[qryscount+tokindex+longest_len_start-1-matchcount]
								< 2){
							printf("One gap lookup kernel- This is not possible\n");
							return;
						} 
						///Debugging
						if (gostart-1-MIN_gap_size-move-matchcount < 0){
							backoff = -1;							
						} else {
							backoff = 
							refstr[gostart-1-MIN_gap_size-move-matchcount];
						}
						
						if (backoff < 2){
							stop = true;//stop this iteration
							flager = false;//stop this ref position
						} else if(backoff == 
								qrysoffsettok[qryscount+tokindex+longest_len_start-1-matchcount]){
							matchcount++;
						} else {
							stop = true;
						}						
					}

					if(matchcount == longest_len_start
						&& 
						checkBoundaryGap(gostart-1-MIN_gap_size-move+1, 
							gostart-1, 
							L_tar, 
							R_tar, 
							RLP)){
						nextpos = atomicAdd(counter, 1);
						oneGapSA[nextpos].position	= blockId;
						oneGapSA[nextpos].str_position = gostart-1-MIN_gap_size-move-longest_len_start+1;
						oneGapSA[nextpos].length = longest_len_end+MIN_gap_size+move+longest_len_start-1;
						//to the end aXb's b	 
					}
				} 

				move++;
				if(longest_len_start+MIN_gap_size+move + longest_len_end > MAX_rule_span){
					flager = false;
				}
			}
		}
		threadx += blockDim.x;
	}

	}

	__global__ void twoGapLookUpSA(	
			int* refstr, 
			//	int* refsa,
			unsigned int toklen,
			int *qrysoffsettok,
			int qryscount,    
			int tokenscount,
			result_t_two* qryresult,
			int* connectoffset,
			int totalconnect,
			result_t* result_connect,
			int* tokindex_qryindex,
			twoGapOnSA* twoGapSA,	
			unsigned int precomp_onegap_count,
			int* frequentList,
			precomp_st_end* precomp_index,
			precompute_enu_3* precomp_onegap,
			unsigned int* counter,
			two_gappy_search* twoGapSearch,
			int distinctTwoGapCount,
			int distinctOneGapCount,
			gappy_search* oneGapSearch,
			oneGapOnSA* oneGapSA,
			unsigned int oneGapSACount,
			unsigned int* RLP,
			uint8_t* L_tar,
			uint8_t* R_tar) {

				unsigned int twoBlockId = blockIdx.y*gridDim.x+blockIdx.x;
				if (twoBlockId >= distinctTwoGapCount){
					return;
				}
				unsigned int oneBlockId = twoGapSearch[twoBlockId].blockid;
				if (oneBlockId >= distinctOneGapCount){
					printf("Not possible inside twoGaponSa kernel! oneId %u | distinctOneGapCount %u\n", oneBlockId, distinctOneGapCount);
					return;
				}

				//No precomputation here
				int startSA = oneGapSearch[oneBlockId].start_on_salist;
				int endSA = oneGapSearch[oneBlockId].end_on_salist;

				if(startSA == -1 && endSA == -1){
					return;
				}

				if (endSA > oneGapSACount || endSA < startSA){
					printf("not possible1 in kernel two gap sa1| endSA %d | oneGapSACount %u | startSA %d\n", endSA, oneGapSACount, startSA);
					return;
				}	

				int threadx = threadIdx.x;
				int dis = endSA - startSA+1;
				int gostart = 0;
				int move = 0;
				bool flager = true;
				int temp = -1;

				//for aXbXc. Add them all minus 1. aXbXc's c starting position.
				//watch for c's longest_end_range
				/*int search_tokindex = oneGapSearch[oneBlockId].qrystart
				  +oneGapSearch[oneBlockId].qrystart_len
				  +oneGapSearch[oneBlockId].gap
				  +oneGapSearch[oneBlockId].qryend_len - 1
				  +twoGapSearch[twoBlockId].gap2;*/
				int search_tokindex = twoGapSearch[twoBlockId].gap2;
				int longest_len_end = twoGapSearch[twoBlockId].qryend_len;
				
				if(longest_len_end!=1){
					printf("Should be one!!\n");
					return;
				}
				
				int nextpos = -1;
				int precomputationIndex = -1;
				int tobesearch_start = 0;

				unsigned int precompstart;
				uint8_t precomplen;
				
				int matchcount;
				int backoff;
				bool stop=false;
				
				int preCache = qrysoffsettok[qryscount+search_tokindex];
				///for debug purpose				
				if (preCache < 2){
					printf("not possible in kernel two gap on SA! query checking wrong\n");
					return;
				}
				///end debug
				
				if(dis == 1 && oneGapSA[startSA].length==0){

					//Precomputation case
					precomputationIndex = oneGapSA[startSA].str_position;
					dis = precomp_index[precomputationIndex].end - 
						precomp_index[precomputationIndex].start + 1;
					tobesearch_start = precomp_index[precomputationIndex].start;

					//check
					if(oneGapSearch[oneBlockId].qrystart_len!=1
							|| oneGapSearch[oneBlockId].qryend_len!=1
							|| precomputationIndex < 0){
						printf("Good to know! Two gap on SA kernel\n");
						return;
					}					

					//precomputation case
					while(threadx < dis){
						precompstart =  precomp_onegap[tobesearch_start+threadx].start;
						precomplen = precomp_onegap[tobesearch_start+threadx].length;
						//last position, not real length
						gostart = precompstart + precomplen;					
						//gostart = oneGapSA[startSA+threadx].str_position+
						//	oneGapSA[startSA+threadx].length;
						move = 0;
						temp = -1;
						flager = true;
						while(flager){
							if(move == 0){
								//Check gap, if the gap is a <2 between two sentences, 
								//then skip
								temp = refstr[gostart+MIN_gap_size];
								if(temp < 2){
									flager = false;
								}
							}
							temp = refstr[gostart+1+MIN_gap_size+move];

							if (precomplen+
									1 + MIN_gap_size+move + 1>MAX_rule_span){
								flager = false;
							}
							
							if (temp < 2 ){
								flager = false;
							} else if (flager && temp == preCache){
								matchcount = 1;
								stop = false;
								while(!stop&&matchcount < longest_len_end
										&& precomplen+matchcount+MIN_gap_size+move + 
										1 + 1 <= MAX_rule_span ){
									////debugging
									if (qrysoffsettok[qryscount+search_tokindex+matchcount]< 2){
										printf("This is not possible - onegap kernel");
										return;
									}
									//debugging
									
									backoff = refstr[gostart+1+MIN_gap_size+move+matchcount];
									if (backoff < 2){
										stop = true;
										flager = false;
									} else if(backoff == 
											qrysoffsettok[qryscount+search_tokindex+matchcount]){
										matchcount++;
									} else {
										stop = true;
									}
								}

								if(matchcount == longest_len_end
									&& checkBoundaryGap(precompstart+precomplen+1, 
											precompstart+1+precomplen+MIN_gap_size+move-1, 
											L_tar, 
											R_tar, 
											RLP)){
									nextpos = atomicAdd(counter, 1);
									twoGapSA[nextpos].position	= twoBlockId;
									twoGapSA[nextpos].str_position = precompstart;
									twoGapSA[nextpos].length = precomplen;// to the end aXb's b	
									twoGapSA[nextpos].length2 = 
										precomplen+
										1+MIN_gap_size+move+longest_len_end-1; 
									//to the end of aXbXc's c, end of c if c has multiple characters.
								}
							} 			
							move++;
						}	
						threadx+=blockDim.x;
					}

				} else {/*if(dis!=1&& oneGapSA[startSA].length!=0){*/
					//None precomputation case
					while(threadx < dis){
						precompstart = oneGapSA[startSA+threadx].str_position;
						precomplen = oneGapSA[startSA+threadx].length;
						///check debug
						if(precomplen==0){
							printf("Not possible!!! inside kernel two gap on sa - length==0\n");
							return;							
						}
						/////				
						gostart = precompstart+precomplen;//last position, not real length
						move = 0;
						temp = -1;
						flager = true;
						while(flager){
							if(move == 0){
								//Check gap, if the gap is a <2 between two sentences, 
								//then skip
								temp = refstr[gostart+MIN_gap_size];
								if(temp < 2){
									flager = false;
								}
							}
							temp = refstr[gostart+1+MIN_gap_size+move];

							if (precomplen+
									1 + MIN_gap_size+move + 1>MAX_rule_span){
								flager = false;
							}
							
							if (temp < 2 ){
								flager = false;
							} else if (flager && temp == preCache){
								matchcount = 1;
								stop = false;
								while(!stop&&matchcount < longest_len_end
										&& precomplen+matchcount+MIN_gap_size+move + 
										1 + 1 <= MAX_rule_span ){
									////debugging
									if (qrysoffsettok[qryscount+search_tokindex+matchcount]< 2){
										printf("This is not possible - onegap kernel");
										return;
									}
									//debugging
									
									backoff = refstr[gostart+1+MIN_gap_size+move+matchcount];
									if (backoff < 2){
										stop = true;
										flager = false;
									} else if(backoff == qrysoffsettok[qryscount+search_tokindex+matchcount]){
										matchcount++;
									} else {
										stop = true;
									}
								}

								if(matchcount == longest_len_end
									&& checkBoundaryGap(precompstart+precomplen+1, 
											precompstart+1+precomplen+MIN_gap_size+move-1, 
											L_tar, 
											R_tar, 
											RLP)){
									nextpos = atomicAdd(counter, 1);
									twoGapSA[nextpos].position	= twoBlockId;
									twoGapSA[nextpos].str_position = precompstart;
									twoGapSA[nextpos].length = precomplen;// to the end aXb's b	
									twoGapSA[nextpos].length2 = 
										precomplen+
										1+MIN_gap_size+move+longest_len_end-1; 
									//to the end of aXbXc's c, end of c if c has multiple characters.
								}
							} 			
							move++;
						}	
						threadx+=blockDim.x;
					}
				}
			}


	__global__ void precomp(
			precompute_enu* oneGapPrecomp,
			int* refstr, 
			int *refsa,
			precompute_enu_2* onegap_precomp,
			unsigned int* counter,
			unsigned int* RLP,
			uint8_t* L_tar,
			uint8_t* R_tar,
			int* featureMissingCount_d) {
		//max_rule_span -> 15; Maximum rule span
		//max_rule_symbols -> 5; Maximum number of symbols (terminals + nontermals) in a rule
		//min_gap_size -> 1; Minimum gap size
		//max_nonterminals -> 2; Maximum number of nonterminals in a rule
		//max_phrase_len -> 4; Maximum frequent phrase length

		int index = blockIdx.x+gridDim.x*blockIdx.y;

		if(index < PRECOMPUTECOUNT*PRECOMPUTECOUNT){	
			featureMissingCount_d[index]=0;
			/*if(index == PRECOMPUTECOUNT*PRECOMPUTECOUNT-1){
			  printf("Got it!\n");
			  }*/
			int tid = threadIdx.x + oneGapPrecomp[index].start;
			//printf("TID %d\n", index);
			unsigned int end = 0;

			unsigned int nextpos;
			int temp;
			int move=0;
			bool flager = true;
			int gostart = -1;

			end = oneGapPrecomp[index].length + oneGapPrecomp[index].start;
			bool reverser = oneGapPrecomp[index].reverse;

			while(tid < end){
				move = 0;
				flager = true;

				if (reverser){
					gostart = refsa[tid];
					if(refstr[gostart]!=oneGapPrecomp[index].token_a  ||
							refstr[refsa[oneGapPrecomp[index].start]] != refstr[gostart]){
						printf("Not possible!!!!!! %d %d\n", refstr[gostart], oneGapPrecomp[index].token_a);
						return;
					}
					while(flager){
						if(move == 0){
							temp = refstr[gostart+MIN_gap_size];
							if(temp < 2){
								flager = false;
							}
						}
						temp = refstr[gostart+1+MIN_gap_size+move];
						if(temp < 2 ){
							flager = false;
						}
						else if (flager && temp == oneGapPrecomp[index].token_b){
							if(checkBoundaryGap(gostart+1, 
									gostart+move+1+MIN_gap_size-1, 
									L_tar, 
									R_tar, 
									RLP)){
								nextpos = atomicAdd(counter, 1);
								if(nextpos > ONEGAP_PRECOMPUT_PREALLOCATION - 1000){
									printf("Stop!!!\n");
									return;
								}
								onegap_precomp[nextpos].index = index;
								onegap_precomp[nextpos].start = gostart;
								//string array position
								//The last position of aXb's b. inclusive
								onegap_precomp[nextpos].length = move+1+MIN_gap_size;
							} else {
								atomicAdd(&(featureMissingCount_d[index]), 1);
							}
						}
						move++;
						if(1+MIN_gap_size+move + 1> MAX_rule_span){
							flager = false;
						}
					}					
				} else {
					gostart = refsa[tid];
					if(refstr[gostart]!=oneGapPrecomp[index].token_b ||
							refstr[refsa[oneGapPrecomp[index].start]] != refstr[gostart]){
						printf("Not possible!!!!!! %d %d\n", refstr[gostart], oneGapPrecomp[index].token_a);
					}
					while(flager){
						if(move == 0 && gostart-MIN_gap_size >= 0){
							temp = refstr[gostart-MIN_gap_size];
							if(temp < 2){
								flager = false;
							}
						}
						if (flager&&gostart-1-MIN_gap_size-move >= 0){						
							temp = refstr[gostart-1-MIN_gap_size-move];
							if(temp < 2 ){
								flager = false;
							}
							else if (flager && temp == oneGapPrecomp[index].token_a){
								if(checkBoundaryGap(gostart-1-MIN_gap_size-move+1, 
										gostart-1, 
										L_tar, 
										R_tar, 
										RLP)){
									nextpos = atomicAdd(counter, 1);
									onegap_precomp[nextpos].index = index;
									onegap_precomp[nextpos].start = gostart-1-MIN_gap_size-move;
									onegap_precomp[nextpos].length = move+1+MIN_gap_size;
								} else {
									atomicAdd(&(featureMissingCount_d[index]), 1);
								}
							}
						} else {
							flager = false;
						}

						move++;
						if(1+MIN_gap_size+move + 1> MAX_rule_span){
							flager = false;
						}
					}
				}
				tid+= blockDim.x;
			}		
			//index += gridDim.x;
			//printf("gridDim %d\n",gridDim.x);
		}	
	}
