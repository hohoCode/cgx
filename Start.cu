#include "hip/hip_runtime.h"
#include "ComTypes.h"
#include "Disk.h"
#include "SuffixArray.h"
#include "Timer.h"
#include "ExtractPair.h"
#include "uthash/uthash.h"
#include "PrintResults.h"

using namespace std;
#define REF_LOAD_BATCH 3
#define PRINTCACHELIMIT 1048576
#define PREALLOCATEDVOCSIZE 500000

static void finalizeRefSet(ref_set_t *refset) {
	int i;
	hashtbl *s, *tmp, *s2;
	hash_intchar *s1, *tmp1, *s3;

	for (i = 0; i < refset->count; i++) {
		hipHostFree(refset->refs[i].buf);
		hipHostFree(refset->refs[i].buf);

		HASH_ITER(hh, refset->refs[i].users, s, tmp) {
			HASH_DEL(refset->refs[i].users, s);
			//free(s->name);
			free(s);
		}
		HASH_ITER(hh, refset->refs[i].intchar, s1, tmp1) {
			HASH_DEL(refset->refs[i].intchar, s1);
			//free(s1->name);
			free(s1);
		}

		HASH_ITER(hh, refset->refs_target[i].users_target, s2, tmp) {
			HASH_DEL(refset->refs_target[i].users_target, s2);
			free(s2);
		}
		HASH_ITER(hh, refset->refs_target[i].intchar_target, s3, tmp1) {
			HASH_DEL(refset->refs_target[i].intchar_target, s3);
			free(s3);
		}
		//free(refset->refs[i].buf);
		//free(refset->refs[i].tokindex);
		//free(refset->refs[i].tokindex_reverse);
	}
	free(refset->refs);
	free(refset->refs_target);
}

void constructQryIndex(qry_set_t * qryset, disk_handler_t * handler, ref_set_t *refset){
	/////Read in queries line by line, and split them into tokens
	/////Record the tokens index, and the starting index inside each queries
	/////And record the number of tokens  

	ref_t * ref = &refset->refs[0];	
	char * line = NULL;
	size_t len = 0;
	int qcount = 0;    
	const char* delim = " ";    
	int count = 0;
	int read = 0;
	int max = -1;
	int percount = 0;

	int* toklen = (int*)malloc(2*sizeof(int)*(handler->qryapplen));
	int* tokindex_qryindex = (int*)malloc(sizeof(int)*(handler->qryapplen));
	if(handler->qryfile == NULL || toklen == NULL)
	{
		printf("not able to open file to construct tokindex\n") ;
		exit(0) ;
	}	
	assert(toklen !=NULL && ref->users != NULL);
	//fprintf(stderr, "Q: ");
	while(read = getline(&line, &len, handler->qryfile)!= -1){        
		qcount++;    
		if (line[len - 1] == '\n'){
			line[len - 1] = '\0';
		}
		char* token = strtok(line, delim);        
		toklen[qcount-1] = count;
		percount = 0;		
		//fprintf(stderr, "Query %d - Count%d\n", qcount-1, count);
		while(token != NULL && !isspace(*token)){            
			percount++;
			struct my_struct *sss = (struct my_struct*)malloc(sizeof(struct my_struct));   
			if(token[strlen(token)-1] == '\n'){
				token[strlen(token)-1] = '\0';
			}
			HASH_FIND_STR(ref->users, token, sss);            
			if(!sss){                                
				/*s = (hashtbl*)malloc(sizeof(hashtbl));                
				  s->id = HASH_COUNT(users) + 1;
				  last = s->id;
				  s->name = (char*)malloc(sizeof(char)*(strlen(token)+1));                
				  strcpy(s->name, token);                
				  HASH_ADD_KEYPTR( hh, users, s->name, strlen(s->name), s);*/
				toklen[handler->qryapplen+count] = -1;
				//fprintf(stderr, "-1 ");
			} else{
				toklen[handler->qryapplen+count] = sss->id;            
				//fprintf(stderr, "%d ", sss->id);
			}		
			tokindex_qryindex[count] = qcount-1;
			//printf("TOKEN: %s\n", token);           
			count++;		  	
			token = strtok(NULL, delim);           
		}
		if (percount > max){
			max = percount;
		}
	}
	fprintf(stderr, "\nMax length of queries is %d\n", max);
	qryset->totaltokens = count;
	qryset->resbufsize = count*sizeof(result_t_two);
	qryset->qryscount = qcount;
	qryset->qrysbufsize = (qryset->totaltokens+qryset->qryscount)*sizeof(int);

	//qryset->qrysbuf = (int*)malloc(qryset->qrysbufsize);
	//qryset->result_two = (result_t_two *)malloc(qryset->resbufsize);
	//qryset->connectoffset = (int*)malloc(qryset->totaltokens*sizeof(int));
	hipHostMalloc((void **)&(qryset->qrysbuf), qryset->qrysbufsize);
	hipHostMalloc((void **)&(qryset->result_two), qryset->resbufsize);
	hipHostMalloc((void **)&(qryset->connectoffset), qryset->totaltokens*sizeof(int));
	hipHostMalloc((void **)&(qryset->tokindex_qryindex), qryset->totaltokens*sizeof(int));

	memcpy(qryset->qrysbuf, toklen, qryset->qryscount*sizeof(int));
	memcpy(qryset->tokindex_qryindex, tokindex_qryindex, qryset->totaltokens*sizeof(int));
	memcpy(qryset->qrysbuf+qryset->qryscount, toklen+handler->qryapplen, qryset->totaltokens*sizeof(int));
	qryset->qrysoffsettok = qryset->qrysbuf;
	free(toklen);	
	free(tokindex_qryindex);
}

void add_intchar(int user_id, char *name, hash_intchar** all) {
	hash_intchar *s = (hash_intchar *)malloc(sizeof(hash_intchar));
	s->id = user_id;
	s->name = (char*)malloc(sizeof(char)*(strlen(name)+1)); 
	memcpy((void*)s->name, name, strlen(name)+1);
	HASH_ADD_INT(*all,id, s);  /* id: name of key field */
}

static void initRefTargetSet(ref_set_t *refset, disk_handler_t *handler, uint	*devicemem) {
	//fprintf(stderr, "\nLoading the target reference\n");
	refset->refs_target = (ref_t_target*)calloc(1, sizeof(ref_t_target));
	assert(refset->refs_target);
	ref_t_target * ref = &refset->refs_target[0];			 
	int* ref_temp = (int*)malloc(sizeof(int)*(handler->tarapplen+ 3));
	int* senidx = (int*)malloc(sizeof(int)*(handler->tarapplen));  ///Number of lines!!
	assert(ref_temp!=NULL);
	assert(senidx != NULL);

	hashtbl *s = NULL;    	
	ref->users_target = NULL;
	ref->intchar_target = NULL;

	int count = 0;    
	int qcount = 0;
	int read = 0;
	const char* delim = " ";    
	char* line = NULL;    
	size_t len = 0;
	int last = -1;
	senidx[0] = 0;
	///Direct Access to the string: from ID to char* pointers
	//ref->vocabulary.push_back(NULL);
	//ref->vocabulary.push_back(NULL);
	char** preVocabulary = (char**)malloc(sizeof(char*)*(PREALLOCATEDVOCSIZE));  //Not working
	while(read = getline(&line, &len, handler->reftargetfile)!= -1){        
		qcount++; 
		size_t newbuflen = strlen(line);
		if (line[newbuflen - 1] == '\n')
			line[newbuflen - 1] = '\0';
		char* token = strtok(line, delim);        
		while(token != NULL && !isspace(*token)){            
			struct my_struct *sss;
			if(token[strlen(token)-1] == '\n'){
				token[strlen(token)-1] = '\0';
			}
			HASH_FIND_STR(ref->users_target, token,sss);            
			if(!sss){                                
				s = (hashtbl*)malloc(sizeof(hashtbl));                
				s->id = HASH_COUNT(ref->users_target) + 2;
				last = s->id;
				s->name = (char*)malloc(sizeof(char)*(strlen(token)+1)); 
				memcpy((void*)s->name, token, strlen(token)+1);
				HASH_ADD_KEYPTR( hh, ref->users_target, s->name, strlen(s->name), s);                
				ref_temp[count] = s->id;
				add_intchar(s->id, token, &ref->intchar_target);
				//ref->vocabulary.(char*)(s->name);
				preVocabulary[s->id] = (char*)s->name;
			} else{
				ref_temp[count] = sss->id;            
			}
			//printf("TOKEN: %s - %d - %d \n", token, strlen(s->name), qcount);           
			count++;           				  	
			token = strtok(NULL, delim);           
		}
		ref_temp[count] = 1;
		count++;
		senidx[qcount]=count;
	}
	///Put this direct access char* array into ref
	if(PREALLOCATEDVOCSIZE<(HASH_COUNT(ref->users_target) + 2)){
		fprintf(stderr, "-->Error: Make sure it is bigger. Check.\n");
		exit(0);
	}
	ref->vocabulary = (char**)malloc(sizeof(char*)*(HASH_COUNT(ref->users_target) + 2));
	memcpy(ref->vocabulary, preVocabulary, sizeof(char*)*(HASH_COUNT(ref->users_target) + 2));
	free(preVocabulary);

	ref_temp[count] = 1;
	count++;
	last++;
	ref_temp[count] = last;
	count++;
	ref->toklen = count;
	fprintf(stderr, "Target Reference toklen number is %d HASH_COUNT TARGET %d INTCHARCOUN %d\n", count, HASH_COUNT(ref->users_target), HASH_COUNT(ref->intchar_target)); 
	long bufsize = count*sizeof(int)+4;
	long intoGPU = count*sizeof(int);
	//fprintf(stderr, "%f MB Memory for target reference will be allocated on GPU\n", (double)intoGPU/(1024*1024));
	//assert(intoGPU < *devicemem);
	//*devicemem -= intoGPU;
	ref->bufsize = bufsize;
	ref->buf = (int*)malloc(ref->bufsize);
	//hipHostMalloc((void **)(&ref->buf),ref->bufsize);

	ref->sentenceind = (int*)malloc(sizeof(int)*(qcount+1));  ///Number of lines!!
	assert(ref->buf);

	ref_temp[count] = ref_temp[count+1] = ref_temp[count+2] = 0;
	memcpy(ref->buf, ref_temp, ref->toklen*sizeof(int));
	memcpy(ref->sentenceind, senidx, (qcount+1)*sizeof(int));

	ref->str = ref->buf;
	free(ref_temp);
	free(senidx);
	//fprintf(stderr, "\nSub target reference count: %d\n",refset->count);	
}

static void initRefSet(ref_set_t *refset, disk_handler_t *handler, uint	*devicemem) {
	int iseof = 0;
	ref_t *prevref = NULL;
	mytimer_t __t;

	refset->refs = (ref_t *)calloc(REF_LOAD_BATCH, sizeof(ref_t));
	assert(refset->refs);
	refset->count = 0;
	timer_start(&__t);
	fprintf(stderr, "\nLoading the reference\n");
	ref_t * ref = &refset->refs[refset->count];
	int* ref_temp = (int*)malloc(sizeof(int)*(handler->refapplen + 3));
	uint8_t * Plong = (uint8_t*)malloc(sizeof(uint8_t)*(handler->refapplen + 3));
	int* senidx = (int*)malloc(sizeof(int)*(handler->refapplen));  ///Number of lines!!
	assert(senidx != NULL);
	assert(Plong != NULL);
	assert(ref_temp!=NULL);

	hashtbl *s = NULL;    
	ref->users = NULL;
	ref->intchar = NULL;

	int count = 0;    
	int qcount = 0;    
	int read = 0;
	const char* delim = " ";    
	char* line = NULL;    
	size_t len = 0;
	int last = -1;
	uint8_t localcount = 0;
	senidx[0] = 0;
	///Direct Access to the string: from ID to char* pointers
	char** preVocabulary = (char**)malloc(sizeof(char*)*(PREALLOCATEDVOCSIZE)); 
	while(read = getline(&line, &len, handler->reffile)!= -1){        
		qcount++;        
		size_t newbuflen = strlen(line);
		if (line[newbuflen - 1] == '\n')
			line[newbuflen - 1] = '\0';
		char* token = strtok(line, delim);        
		localcount = 0;
		while(token != NULL && !isspace(*token)){            
			struct my_struct *sss;    
			if(token[strlen(token)-1] == '\n'){
				token[strlen(token)-1] = '\0';
			}
			HASH_FIND_STR(ref->users, token,sss);            
			if(!sss){                                
				s = (hashtbl*)malloc(sizeof(hashtbl));                
				s->id = HASH_COUNT(ref->users) + 2;
				last = s->id;

				s->name = (char*)malloc(sizeof(char)*(strlen(token)+1)); 
				memcpy((void*)s->name, token, strlen(token)+1);
				HASH_ADD_KEYPTR( hh, ref->users, s->name, strlen(s->name), s);                
				ref_temp[count] = s->id;
				add_intchar(s->id, token, &ref->intchar);
				preVocabulary[s->id] = (char*)s->name;
			} else{
				ref_temp[count] = sss->id;            
			}
			Plong[count] = localcount;
			localcount++;
			//printf("TOKEN: %s - %d - %d\n", token, strlen(s->name), f);           
			count++;           				  	
			token = strtok(NULL, delim);           
		}
		ref_temp[count] = 1;
		Plong[count] = 0;
		count++;
		senidx[qcount]=count;
	}
	///Put this direct access char* array into ref
	//Security check
	if(PREALLOCATEDVOCSIZE<(HASH_COUNT(ref->users) + 2)){
		fprintf(stderr, "-->Error: Make sure it is bigger. Check.\n");
		exit(0);
	}
	ref->vocabulary = (char**)malloc(sizeof(char*)*(HASH_COUNT(ref->users) + 2));
	memcpy(ref->vocabulary, preVocabulary, sizeof(char*)*(HASH_COUNT(ref->users) + 2));
	free(preVocabulary);

	ref_temp[count] = 1;
	Plong[count] = 0;
	count++;
	last++;
	ref_temp[count] = last;
	count++;
	ref->toklen = count;
	fprintf(stderr, "Reference toklen number is %d HASH_COUNT %d INTCHARCOUN %d\n", count, HASH_COUNT(ref->users), HASH_COUNT(ref->intchar)); 

	size_t bufsize = count*4*sizeof(int)+4;
	size_t sa_size = count*sizeof(int);
	size_t Psize = count*sizeof(uint8_t);
	size_t intoGPU = count*4*sizeof(int);
	fprintf(stderr, "%f MB Memory for reference will be allocated on GPU\n", (double)bufsize*4/(6*1024*1024));
	assert(intoGPU < *devicemem);
	*devicemem -= intoGPU;
	ref->bufsize = bufsize;
	ref->distinctTokenCount = HASH_COUNT(ref->users) + 2;

	/////Memory Allocation buf and sa separation!
	hipHostMalloc((void **)(&ref->P), Psize);
	hipHostMalloc((void **)(&ref->buf), bufsize);
	hipHostMalloc((void **)(&ref->sa),sa_size);
	hipHostMalloc((void **)(&ref->str),sa_size);

	/*ref->P = (uint8_t*)malloc(Psize);	
	  ref->buf = (int*)malloc(bufsize);
	  ref->sa = (int*)malloc(sa_size);
	  ref->str = (int*)malloc(sa_size);*/

	ref->sentenceind = (int*)malloc(sizeof(int)*(qcount+1));  ///Number of lines!!

	assert(ref->buf);
	ref_temp[count] = ref_temp[count+1] = ref_temp[count+2] = 0;
	memcpy(ref->str, ref_temp, ref->toklen*sizeof(int));
	memcpy(ref->P, Plong, ref->toklen*sizeof(uint8_t));
	memcpy(ref->sentenceind, senidx, (qcount+1)*sizeof(int));

	suffixArrayConstruct(ref, last, ref_temp);
	ref->sentence_count = qcount;

	free(ref_temp);
	free(Plong);
	free(senidx);
	/* Increment the count and resize the ref set
	   array in case we ran out of space */
	refset->count++;
	if ((refset->count % REF_LOAD_BATCH) == 0) {
		refset->refs = (ref_t *)realloc(refset->refs, 
				sizeof(ref_t) * 
				(refset->count + REF_LOAD_BATCH));
		assert(refset->refs);
	}
	prevref = &refset->refs[refset->count - 1];

	timer_stop(&__t);
	/*fprintf(stderr, "\nSub reference count: %d\n"
	  "Loading the reference completed: %f Sec, HASHCOUNT REF %d INTCHAR COUNT %d\n", 
	  refset->count, timer_elapsed(&__t)/1000.0, HASH_COUNT(users), HASH_COUNT(intchar));    */
}

static void 
finalizeQrySetBuffers(qry_set_t *qryset, 
		int refsetcount) {	

	hipHostFree(qryset->result_two);
	hipHostFree(qryset->qrysbuf);
	hipHostFree(qryset->connectoffset);
	hipHostFree(qryset->result_connect);
}

void recordTime(options_t * options,
		ref_set_t *refset,
		timing_t * timing, 
		float overall) {

	int i = 0;

	double total = 
		timing->suffixarray +
		timing->qrysin +
		timing->refsin +
		timing->resout +
		timing->kernel +
		timing->printout +
		timing->kernel2 + 
		timing->extractin +
		timing->extractkernel;

	if (options->timefile) {
		FILE * fh = fopen(options->timefile, "a");
		fprintf(fh, 
				"total: %f , "
				"kernel 1: %f , "
				/*"kernel 2: %f , "*/
				"printmatches: %f , "
				"querystoGPU: %f , "
				"outputfromGPU: %f , "
				"referencetoGPU: %f , "
				"queriesfromdisk: %f , "
				"refpreprocessing: %f , "
				//"reflen: %d , "
				//"subreflen: %d , "
				"subrefcount: %d , "
				"subrefsize: %d , "
				"minmatchlen: %d\n",
				total,
				timing->kernel,
				//timing->kernel2,
				timing->printout,
				timing->qrysin,
				timing->resout,
				timing->refsin,
				timing->qrysload,
				timing->suffixarray,
				//reflen,
				/*refset->refs[0].len,*/
				refset->count,
				refset->refs[0].bufsize,
				options->minmatchlen);
		fclose(fh);
	}

	/* print the results again to stderr */
	fprintf(stderr, 
			"\n\ntotal: %f , "
			"kernel 1: %f , "
			/*"kernel 2: %f , "*/
			"printmatches: %f , "
			"querystoGPU: %f, "
			"outputfromGPU: %f, "
			"referencetoGPU: %f, "
			"queriesfromdisk: %f, "
			"refpreprocessing: %f, "
			"extraction transfer: %f, "
			"extraction kernel: %f\n",
			total,
			timing->kernel,
			//timing->kernel2,
			timing->printout,
			timing->qrysin,
			timing->resout,
			timing->refsin,
			timing->qrysload,
			timing->suffixarray,
			timing->extractin,
			timing->extractkernel);

}

int getDeviceMemory() {


	hipDeviceProp_t props;
	int device;
	hipError_t err;

	hipGetDevice(&device);
	//assert(err == hipSuccess);

	hipGetDeviceProperties(&props, device);
	//assert(err == hipSuccess);

	return int((double(props.totalGlobalMem) * 95.0/100.0));

}

void 
start(options_t * options) {

	disk_handler_t handler = DISK_INIT_HANDLER;
	ref_set_t refset;
	qry_set_t qryset;


	uint devicemem = 2000*1024*1024;//getDeviceMemory();
	//hipDeviceSetLimit(hipLimitPrintfFifoSize, PRINTCACHELIMIT);
	//fprintf(stderr, "Print cache limit is %f MB\n", (double)PRINTCACHELIMIT/(1024*1024));

	mytimer_t __t, __overall,__overall_sub;
	timing_t timing = {
		0,
		0,
		0,
		0,
		0,
		0,
		0
	};

	/* REF_DEV_BUF_RATIO is the ratio of device memory that 
	   will be used for the reference, the rest is for queries */
	int maxreflen = 
		suffixArrayGetEquivalentMaxRefLen(int(double(devicemem)*REF_DEV_BUF_RATIO),
				options->fingerlen);

	timer_start(&__overall);
	int rc = 
		diskInit(&handler,
				options->reffile,
				options->qryfile,
				maxreflen,
				options->reftargetfile,
				options->align, 
				options->wordscdec,
				options->destinationDirectory);

	if (!rc) {
		return;
	}

	fprintf(stderr, 
			"\nAvailable Device Memory: %.3fMB\n"
			"Approximate Reference Length: %.3fMB\n"
			"Approximate Query Length: %d\n"
			"Sub Reference Length: %.3fMB\n", 
			(double)devicemem / (1024.0*1024.0),
			(double)handler.refapplen / (1024.0*1024.0),
			handler.qryapplen,
			(double)handler.refmaxlen / (1024.0*1024.0));

	timer_start(&__t);

	///Read Reference Text In, Construct the SA Array
	initRefSet(&refset, &handler, &devicemem);
	initRefTargetSet(&refset, &handler, &devicemem);

	//map<string, categ> word_score = initWordPossibility(&handler, users, users_target);
	//Already cuda memory allocated.
	lexicalFileCuda* cudaLexFile = initWordPossibilityIntKey(&handler, 	&refset);

	timing.suffixarray = timer_stop(&__t);

	//fprintf(stderr, "\nLoading query set\n");		
	timer_start(&__overall_sub);
	timer_start(&__t);
	///Read Qryset In
	constructQryIndex(&qryset, &handler, &refset);
	timing.qrysload += timer_stop(&__t);
	//fprintf(stderr, "\nLoading query complete\n");

	//////////Reading the Alignment Files - This is an one-time offline operation, so that timings can be deducted
	timer_start(&__t);
	initAlignment(&refset, &handler, &devicemem);		
	double alignment_deduction = timer_stop(&__t); ////

	void * ctx = suffixArraySearchInit(
			refset.refs[0].toklen*sizeof(int)*2, 
			qryset.qrysbufsize, 
			qryset.totaltokens*sizeof(int), 
			refset.refs[0].toklen*sizeof(int));
	int iseof = 0;
	int flag = 0;
	int twoway = 1;
	/* Actual matching */
	suffixArraySearch(ctx, &refset, &qryset, 1, &timing, twoway);
	/* Print out the results */
	timer_start(&__t);
	//printResults_two_file(&refset, &qryset, options, users, intchar);
	timing.printout += timer_stop(&__t);
	/////////////////////////////////// Extract Pair
	fprintf(stderr, "Start Extract Pair\n");
	//////////////////////////////////////////////
	suffixArraySearchFinalize_One(ctx);////Array Cleanup - timings are included.
	ExtractPairs_Large_Data_Gappy(ctx, 
			&refset, 
			&qryset,
			&timing, 
			cudaLexFile,
			&handler);		//Changed to new function with gappy.
	///////////////////////////////////
	//fprintf(stderr, "Finish Extract Pair\n");

	timer_stop(&__overall_sub);
	timer_stop(&__overall);
	/*fprintf(stderr, 
			"Excluding Preprocessing Time - Qry Loading&Construction/Suffix Extraction/Consistent Pairs Extraction/Pair Scoring&Counting/All GPU Costs/All Rules PrintOut/Disk IO %f - TOTAL %f\n", 
			timer_elapsed(&__overall_sub)-alignment_deduction, timer_elapsed(&__overall)-alignment_deduction);*/

	print_query_GPU_Gappy(
			qryset.continousQueryWithID,		
			qryset.oneGapQueryWithID,
			qryset.twoGapQueryWithID,
			qryset.qryscount,
			qryset.globalOnPairsUpDownContinous,
			qryset.globalOnPairsUpDownGappy,
			qryset.globalOnPairsUpDownTwoGap,
			qryset.fast_speed,
			qryset.fast_speed_one_gap,
			qryset.fast_speed_two_gap,
			options->destinationDirectory,
			qryset.distinctContinousCount,//continous noGapsearch count
			qryset.distinctOneGapCount,//oneGapSearch count
			qryset.distinctTwoGapCount);//TwoGapSearch count
	/*print_query_GPU_Continous(
	  qryset.continousQueryWithID, 
	  qryset.qryscount,
	  qryset.globalOnPairsUpDownContinous,
	  qryset.fast_speed,
	  options->destinationDirectory);*/
	//recordTime(options, &refset, &timing, timer_elapsed(&__overall));

	// clean up!
	extractPairFinalize(ctx);
	suffixArraySearchFinalize(ctx);
	finalizeQrySetBuffers(&qryset, refset.count);
	finalizeRefSet(&refset);
	diskFinalize(&handler);
}



