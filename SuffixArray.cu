#include "hip/hip_runtime.h"
#include "ComTypes.h"
#include "Timer.h"
#include <thrust/device_vector.h>
#include <thrust/sort.h>
//#include <thrust/host_vector.h>
#include <thrust/version.h>

#include "GappyLook.h"

#define ISSPACE(x) ((x) == ' ' || (x) == '\t' || (x) == '\n')
#define cache 10
#define ISPUNCSPA(x) ((x) == ' ' || (x) == '\t' || (x) == '.' || (x) == '?'  || (x) == ','  || (x) == '!'  || (x) == ':'  || (x) == ';'  || (x) == '}'  || (x) == ']'  || (x) == ')'  || (x) == '*'  || (x) == '@' || (x) == '\n' || (x) == '"' || (x) == '\'')

#define COMP1(s1, s2, newb, mymatchcount, qrylimit, ok)		\
	mymatchcount = 0;\
ok = 0;\
do {									\
	int ref = *(newb+ s1 + mymatchcount);				\
	int qry = *(s2 + mymatchcount); 					\
	while (mymatchcount < qrylimit && (ref ==qry) && ref != 1 && qry != -1) {\
		mymatchcount++;\
		ref = *(newb+ s1 + mymatchcount);\
		qry = *(s2 + mymatchcount);\
	}\
	if (qry==-1 || mymatchcount == qrylimit){\
		ok = 1;\
	}\
	/*printf("str %c - q %c - myrc %d - count %d - saind %d\n",a, b, myrc, mymatchcount, s1);*/\
} while(0)

	struct twoGapEnumerationCompare
	{
		__host__ __device__
			bool operator()(const gapPattern2& a, const gapPattern2& b) const
			{
				if(a.blockid != b.blockid){
					return (a.blockid < b.blockid);
				} else if(a.number != b.number){
					return (a.number<b.number); 
				} else {
					for(int i=0; i<b.number;i++){
						if(a.pattern[i]!=b.pattern[i]){
							return a.pattern[i] < b.pattern[i];
						}
					}
				}
				return true;
			}
	};

	struct oneGapEnumerationCompare
	{
		__host__ __device__
			bool operator()(const gapPattern1& a, const gapPattern1& b) const
			{
				if(a.number != b.number){
					return (a.number<b.number); 
				} else {
					for(int i=0; i<b.number;i++){
						if(a.pattern[i]!=b.pattern[i]){
							return a.pattern[i] < b.pattern[i];
						}
					}
				}
				return true;
			}
	};

	struct oneGapSACompare
	{
		__host__ __device__
			bool operator()(const gappytyp_sa& a, const gappytyp_sa& b) const
			{
				return (a.position < b.position) ||((a.position == b.position) 
					&& a.str_position < b.str_position);
			}
	};

	struct twoGapSACompare
	{
		__host__ __device__
			bool operator()(const gappytyp_sa2& a, const gappytyp_sa2& b) const
			{
				return (a.position < b.position)||
					((a.position == b.position) && a.str_position < b.str_position);
			}
	};

	struct twoGapResCompare
	{
		__host__ __device__
			bool operator()(const rule_twogap& a, const rule_twogap& b) const
			{
				return (a.twogappyindex < b.twogappyindex);
			}

	};

	struct oneGapResCompare
	{
		__host__ __device__
			bool operator()(const rule_onegap& a, const rule_onegap& b) const
			{
				return (a.gappy_index < b.gappy_index);
			}

	};

__global__ void suffixArrayFindConnectionTwoWayTDI(
		int* refstr, 
		int *refsa,
		/*int *reflcp,*/
		int * reflcpleft,
		int * reflcpright,
		int reflen,
		int *qrysoffsettok,
		int qryscount,    
		int tokenscount,
		result_t_two* result,
		int* connectoffset,
		int totalconnect,
		result_t* result_connect) {

	int tokindex = threadIdx.x + blockDim.x * blockIdx.x;
	if(tokindex >= tokenscount){
		return;
	}
	int longestmatch = result[tokindex].longestmatch ;
	if (longestmatch <=1){
		return;
	}
	int start = connectoffset[tokindex];		
	if(start < 0 ){
		return;
	}
	int seq = blockIdx.y/2;
	int up = blockIdx.y%2;		

	int match = 2 + seq;
	start += seq;
	if (match > longestmatch){
		return;
	}
	int L, R, M, Rlcp = 0, skip = 0, Llcp = 0;
	int startREF = -1;
	int temp = -1;
	int longest = -1;
	int longlen = -1;
	int firstfindhit = -1;
	int firstfindhitL = -1;		
	int firstfindhitR = -1;		
	int firstfindhitlen = -1;
	int holdtemp = -1;		
	int foundexactlcp = 0;
	int a, b;
	int* query = qrysoffsettok + qryscount+ tokindex;
	int LL = result[tokindex].firstfindhitL;
	int MM = result[tokindex].firstfindhit;
	int RR = result[tokindex].firstfindhitR;

	while(match <= longestmatch){
		L = LL;
		R = RR;
		//////////////////////////////////////////////////////////////////////////////////////
		startREF = -1;
		temp = -1;
		holdtemp = -1;
		longlen = 0;
		foundexactlcp = 0;
		firstfindhit = -1;
		firstfindhitL = -1;
		firstfindhitR = -1;
		Llcp = 0;
		Rlcp = 0;
		while (R - L > 1) {
			longlen = 0;
			if(L == LL && R == RR){
				M = MM;
			} else {
				M = (L + R) >> 1;
			}
			//printf("#Tokindex %d - L %d - R %d - M%d\n", tokindex, L, R, M);
			if (Llcp >= Rlcp){
				longlen = Llcp;
				if (L == M-1){
					skip = reflcpleft[M]; 
				} else{
					holdtemp = (L + M)>>1;
					skip = fminf(reflcpleft[holdtemp], reflcpright[holdtemp]);
				}

				if(longlen < skip){
					L = M;
				} else if (longlen > skip) {
					R = M;
					Rlcp = skip;
				} else {					
					//printf("Comparison LlCP/SKIP %d\n", skip);
					startREF = refsa[M] + longlen;
					a = *(query + longlen);
					b = refstr[startREF];
					if (a == -1) {
						foundexactlcp = 1;
						printf("THIS IS NOT POSSIBLE!!!!\n");                                             	
						break;
					}
					//printf("TBD Llcp %d - REF %d - Tokindex %d - Qryindex %d - qry%c - str%c\n", longlen, startREF, tokindex, qryindex, a, b);							
					if(a!= -1 && b!=1 ){
						temp = a - b;
						while (a != -1 && b !=1 && temp == 0) {									
							longest = M;
							//printf("Middle Matched  Llcp %d - REF %d - Tokindex %d - Qryindex %d - qry%c - str%c\n", longlen, startREF, tokindex, qryindex, a, b);
							longlen++;
							/*if (longlen >= toklen - tokindex) {									
							  foundexactlcp = 1;
							  break;
							  }*/
							startREF++;

							if(firstfindhit == -1 && M >=0 && longlen >= match){
								firstfindhit = M;
								firstfindhitL = L;
								firstfindhitR = R;
								firstfindhitlen = longlen;
								foundexactlcp = 1;
								break;
							}
							a = *(query + longlen);
							b = refstr[startREF];
							if (a == -1) {									
								foundexactlcp = 1;
								printf("THIS IS NOT POSSIBLE!!!!\n");          
								break;
							}
							if(a != -1 && b !=1){
								temp = a - b;
							}
						}
						if (foundexactlcp == 1){
							//printf("Can you believe it? Tokindex%d - match%d\n", tokindex, match);
							break;
						}
					}

					if (a == -1){
						R = M;
						L = M;//break;
					} else if (b ==1) {
						L = M;
						Llcp = longlen;
					} else if (temp > 0){
						L = M;
						Llcp = longlen;
					} else {
						R = M;
						Rlcp = longlen;
					}
				}

			}else {
				longlen = Rlcp;
				if (R == M+1){
					skip = reflcpright[M]; 
				} else{
					holdtemp = (R + M)>>1;
					skip = fminf(reflcpleft[holdtemp], reflcpright[holdtemp]);
				}

				//printf("Rlcp Iteration - R %d - L %d - M %d - Llcp %d - Rlcp %d - M'M %d - TID %d - Qryindex %d\n", R, L, M, Llcp, Rlcp, skip, tokindex, qryindex);
				if(longlen < skip){
					R = M;                        
				} else if (longlen > skip) {
					L = M;
					Llcp = skip;
				} else {					
					//printf("Comparison LlCP/SKIP %d\n", skip);
					startREF = refsa[M] + longlen;	
					a = *(query + longlen);
					b = refstr[startREF];
					if (a == -1) {									
						foundexactlcp = 1;
						break;
					}
					if(a!= -1 && b!=1){
						temp = a - b;					
						while (a != -1 && b !=1 && temp == 0) {									
							longest = M;
							//printf("Middle Matched Ref = %c - Llcp %d - REF %d - tokindex %d - qryindex %d\n", *(refstr+startREF), longlen, startREF, tokindex, qryindex);
							longlen++;
							startREF++;

							if(firstfindhit == -1 && M >=0 && longlen >= match){
								firstfindhit = M;
								firstfindhitL = L;
								firstfindhitR = R;
								firstfindhitlen = longlen;
								foundexactlcp = 1;
								break;
							}
							a = *(query + longlen);
							b = refstr[startREF];
							if (a == -1) {									
								foundexactlcp = 1;
								printf("THIS IS NOT POSSIBLE!!!!\n"); 
								break;
							}	
							if(a != -1  && b !=1){
								temp = a - b;
							}
						}
						if (foundexactlcp == 1){
							//printf("Can you believe it? Tokindex%d - match%d\n", tokindex, match);
							break;
						}
					}

					if (a == -1){
						R = M;
						L = M;//break;
					} else if (b ==1) {
						L = M;
						Llcp = longlen;
					} else if (temp > 0){
						L = M;
						Llcp = longlen;
					} else {
						R = M;
						Rlcp = longlen;
					}
				}

			}
		}

		if (firstfindhit == -1){
			printf("Firstfindhit is -1 - THIS NOT POSSIBLE!!!!\n %d - not possible!!!! longestmatch %d - totalcon %d - match%d\n", tokindex, longestmatch, totalconnect, match); 
			break;
		}

		if (firstfindhit != -1 && longlen > 0 && foundexactlcp == 1){
			/// Here we have found something, this is the final step
			/// We are going to locate the upper bound, and lower bound - Final step
			///Now I have L R and longest.					
			///Deal with upper and lower bound!
			longest = firstfindhit;			
			//printf("QID %d - LongLEN %d - FirstFINDHIT %d\n", qryindex, longlen, longest); 
			if (up == 1){
				R = firstfindhit;
				L = firstfindhitL;
				while (R - L > 1){
					M = (L + R) >> 1;
					holdtemp = (R + M)>>1;
					if (R == M+1){
						skip = reflcpright[M]; 
					} else{
						skip = fminf(reflcpleft[holdtemp], reflcpright[holdtemp]); //LCP(M, M')
					}
					//printf("!! Matched= R%d - L%d - M%d - LCP(R,M)%d\n", R, L, M, skip);
					if (skip >= match){
						longest = M;
						R = M;
						firstfindhitlen = skip;
					} else {
						L = M;
					}
				}			   	
				////Write results
				result_connect[start].up = longest;
			} else {
				R = firstfindhitR;
				L = firstfindhit;
				while (R - L > 1){
					M = (L + R) >> 1;
					holdtemp = (L + M)>>1;

					if (L == M-1){
						skip = reflcpleft[M]; 
					} else{
						skip = fminf(reflcpleft[holdtemp], reflcpright[holdtemp]); //LCP(M, M')
					}

					//printf("!! Matched= R%d - L%d - M%d - LCP(L,M)%d\n", R, L, M, skip);
					if (skip >= match){
						longest = M;
						L = M;
						firstfindhitlen = skip;
					} else {
						R = M;
					}
				}
				////Write results
				result_connect[start].down = longest;
			}
		}

		//////////////////////////////////////////////////////////////////////////////////////
		match += gridDim.y/2;
		start += gridDim.y/2;
	}
}

__global__ void suffixArrayFindLwRwKernelTwoWayTDI(
		int* refstr, 
		int *refsa,
		/*int *reflcp,*/
		int * reflcpleft,
		int * reflcpright,
		int reflen,
		int *qrysoffsettok,
		int qryscount,    
		int tokenscount,
		result_t_two*result) {

	int qryindex = threadIdx.y + blockDim.y * blockIdx.y;
	int tokindex = threadIdx.x;

	if (qryindex >= qryscount ){
		return;
	}

	int queryoffset = qrysoffsettok[qryindex];
	int toklen = -1;
	if (qryindex == qryscount - 1){
		toklen = tokenscount - queryoffset;
	} else {
		toklen = qrysoffsettok[qryindex+1] - queryoffset;
	}
	int up = blockIdx.x % 2;

	//printf("NOeeD %d - toklen %d - QID %d\n", tokindex, toklen, qryindex);
	if (tokindex >= toklen){
		//printf("EXCEED %d - toklen %d - QID %d\n", tokindex, toklen, qryindex);
		return;
	}


	//char *query = &(qrys[qrysoffset[qryindex]]);
	int L, R, M, Rlcp = 0, skip = 0, Llcp = 0;
	int startREF = -1;
	int temp = -1;
	int longest = -1;
	int firstfindhit = -1;
	int firstfindhitL = -1;		
	int firstfindhitR = -1;		
	int firstfindhitlen = -1;
	int holdtemp = -1;
	int longlen = -1;
	int foundexactlcp = 0;
	int ok = 0;
	int a, b;
	result += queryoffset;		

	//while ( tokindex < toklen ){
	//__syncthreads();
	if(result[tokindex].longestmatch >0&& up == 1){
		result[tokindex].longestmatch = 0;
	}
	//printf("tokindex %d - toklen %d - iteration %d - qryindex %d - reflen %d\n", tokindex, toklen, iteration, qryindex, reflen);
	a = qrysoffsettok[qryscount+ queryoffset + tokindex];  
	if(a == -1){
		return;
		//goto END;
		/*iteration++;
		  tokindex += iteration*blockDim.x;
		  continue;*/
	}

	L = 0;
	R = reflen - 1;
	Llcp = 0;
	Rlcp = 0;

	/**First check the reference bundary see whether there is a match or not*/
	/**Starting from Right side */
	foundexactlcp = 0;			
	longest = -1;
	firstfindhit = -1;			
	firstfindhitL = -1;		
	firstfindhitR = -1;		
	firstfindhitlen = -1;

	longlen = -1;
	ok = 0;
	COMP1(refsa[R], qrysoffsettok +qryscount+ queryoffset + tokindex , refstr, Rlcp, toklen-tokindex, ok);//Ref -rc- Query
	if (Rlcp >0 && ok == 1){			
		foundexactlcp = 1; ///Found longest!
		longest = R;
		longlen = Rlcp;
		if (up != 1){
			result[tokindex].longestmatch = Rlcp;
			//result[tokindex].longestind = R;//First Longest
		}
	}
	//printf("RIGHT UP %d - up/down %d - Rlcp %d - dingdong %d - exactlcp %d - qryindex %d\n", up, R, Rlcp, dingdong, exactlcp, qryindex);

	if (Rlcp > 0){
		firstfindhit = R;
		firstfindhitL = L;
		firstfindhitR = R;
		firstfindhitlen = Rlcp;
		if (up != 1){
			result[tokindex].down = R;
			//result[tokindex].lcpdown = Rlcp;					
			//goto END;
			return;
			//iteration++;
			//tokindex += iteration*blockDim.x;
			//printf("UP %d - up/down %d - Rlcp %d\n", up, R, Rlcp);
			//continue;
		} else {
			result[tokindex].up = R;
			//result[tokindex].lcpup = Rlcp;
		}
	}

	/****Now move to the left side, check whether there is a match or not*/
	ok = 0;			

	/***If we have found the longest match in the bundary, we do not need to find another one in the middle*/
	/***If we have not found any match in the two bundary, we need cotinue to find a hit in the middle*/
	/***Once there is a find on the longest match, we break and move to the next step*/
	if(foundexactlcp == 0 ){
		// Look for Lw
		/** This is using binary seearch for finding a longest match in the middle*/
		/**Binary seach with non-standard LCP*/
		startREF = -1;
		temp = -1;
		holdtemp = -1;
		longlen = 0;
		int* query = qrysoffsettok + qryscount+ queryoffset + tokindex;  
		while (R - L > 1) {
			longlen = 0;
			M = (L + R) >> 1;
			//printf("#Tokindex %d - L %d - R %d - M%d\n", tokindex, L, R, M);
			if (Llcp >= Rlcp){
				longlen = Llcp;
				if (L == M-1){
					skip = reflcpleft[M]; 
				} else{
					holdtemp = (L + M)>>1;
					skip = fminf(reflcpleft[holdtemp], reflcpright[holdtemp]);
				}

				//printf("Llcp Iteration - R %d - L %d - M %d - Llcp %d - Rlcp %d - MM' %d - TID %d - qrydindex %d\n", R, L, M, Llcp, Rlcp, skip, tokindex, qryindex);
				if(longlen < skip){
					L = M;
				} else if (longlen > skip) {
					R = M;
					Rlcp = skip;
				} else {					
					//printf("Comparison LlCP/SKIP %d\n", skip);
					startREF = refsa[M] + longlen;
					a = *(query + longlen);
					b = refstr[startREF];
					if (longlen >= toklen - tokindex || a == -1) {
						foundexactlcp = 1;
						//printf("BREAK QID %d TID %d\n", qryindex, tokindex);                                    
						break;
					}
					//printf("TBD Llcp %d - REF %d - Tokindex %d - Qryindex %d - qry%c - str%c\n", longlen, startREF, tokindex, qryindex, a, b);							
					if(a!= -1 && b!=1 ){
						temp = a - b;
						while (a != -1 && b !=1 && temp == 0) {									
							longest = M;
							//printf("Middle Matched  Llcp %d - REF %d - Tokindex %d - Qryindex %d - qry%c - str%c\n", longlen, startREF, tokindex, qryindex, a, b);
							longlen++;
							startREF++;

							if(firstfindhit == -1 && M >=0){
								firstfindhit = M;
								firstfindhitL = L;
								firstfindhitR = R;
							}
							if (firstfindhit == M){
								firstfindhitlen = longlen;
							}
							//printf("Middle Matched  Llcp %d - REF %d - Tokindex %d - Qryindex %d - qry%c - str%c - 2nd - longlen %d - exactlcp %d\n", longlen, startREF, tokindex, qryindex, a, b, longlen, exactlcp);
							if (longlen >= toklen - tokindex) {									
								foundexactlcp = 1;
								//printf("BREAK QID %d TID %d\n", qryindex, tokindex);                                    
								break;
							}
							a = *(query + longlen);
							b = refstr[startREF];
							if (a == -1){
								foundexactlcp = 1;									
								break;
							}	
							if(a != -1 && b !=1){
								temp = a - b;
							}
						}
						if (foundexactlcp == 1){
							break;
						}
					}

					if (a == -1){
						R = M;
						L = M;//break;
					} else if (b ==1) {
						L = M;
						Llcp = longlen;
					} else if (temp > 0){
						L = M;
						Llcp = longlen;
					} else {
						R = M;
						Rlcp = longlen;
					}
				}

			}else {
				longlen = Rlcp;
				if (R == M+1){
					skip = reflcpright[M]; 
				} else{
					holdtemp = (R + M)>>1;
					skip = fminf(reflcpleft[holdtemp], reflcpright[holdtemp]);
				}

				//printf("Rlcp Iteration - R %d - L %d - M %d - Llcp %d - Rlcp %d - M'M %d - TID %d - Qryindex %d\n", R, L, M, Llcp, Rlcp, skip, tokindex, qryindex);
				if(longlen < skip){
					R = M;                        
				} else if (longlen > skip) {
					L = M;
					Llcp = skip;
				} else {					
					//printf("Comparison LlCP/SKIP %d\n", skip);
					startREF = refsa[M] + longlen;	
					a = *(query + longlen);
					b = refstr[startREF];
					if (longlen >= toklen - tokindex || a == -1) {									
						foundexactlcp = 1;
						//printf("BREAK QID %d TID %d\n", qryindex, tokindex);                                    
						break;
					}
					if(a!= -1 && b!=1){
						temp = a - b;					
						while (a != -1 && b !=1 && temp == 0) {									
							longest = M;
							//printf("Middle Matched Ref = %c - Llcp %d - REF %d - tokindex %d - qryindex %d\n", *(refstr+startREF), longlen, startREF, tokindex, qryindex);
							longlen++;
							startREF++;

							if(firstfindhit == -1 && M >=0){
								firstfindhit = M;
								firstfindhitL = L;
								firstfindhitR = R;
							}
							if (firstfindhit == M){
								firstfindhitlen = longlen;
							}
							if (longlen >= toklen - tokindex) {									
								foundexactlcp = 1;
								//printf("BREAK QID %d TID %d\n", qryindex, tokindex);                                    
								break;
							}				
							a = *(query + longlen);
							b = refstr[startREF];
							if (a == -1){
								foundexactlcp = 1;									
								break;
							}				
							if(a != -1  && b !=1){
								temp = a - b;
							}
						}
						if (foundexactlcp == 1){
							break;
						}
					}

					if (a == -1){
						R = M;
						L = M;//break;
					} else if (b ==1) {
						L = M;
						Llcp = longlen;
					} else if (temp > 0){
						L = M;
						Llcp = longlen;
					} else {
						R = M;
						Rlcp = longlen;
					}
				}

			}
		}
	}

	if(longlen >0 && up == 1){
		result[tokindex].longestmatch = longlen;
		//result[tokindex].longestind = longest;
	}
	if (firstfindhit == -1 && longlen >  0){
		printf("THIS FIRSTHIT NOT POSSIBLE!\n");
	}

	if (firstfindhit != -1 && longlen > 0){
		/// Here we have found something, this is the final step
		/// We are going to locate the upper bound, and lower bound - Final step
		///Now I have L R and longest.					
		///Deal with upper and lower bound!
		longest = firstfindhit;
		if(up == 1){
			result[tokindex].firstfindhit = firstfindhit;
			result[tokindex].firstfindhitL = firstfindhitL;
			result[tokindex].firstfindhitR = firstfindhitR;
		}

		//printf("QID %d - LongLEN %d - FirstFINDHIT %d\n", qryindex, longlen, longest); 
		if (up == 1){
			R = firstfindhit;
			L = firstfindhitL;
			while (R - L > 1){
				M = (L + R) >> 1;
				holdtemp = (R + M)>>1;
				if (R == M+1){
					skip = reflcpright[M]; 
				} else{
					skip = fminf(reflcpleft[holdtemp], reflcpright[holdtemp]); //LCP(M, M')
				}
				//printf("!! Matched= R%d - L%d - M%d - LCP(R,M)%d\n", R, L, M, skip);
				if (skip >= 1){
					longest = M;
					R = M;
					//firstfindhitlen = skip;
				} else {
					L = M;
				}
			}			   	
			////Write results
			result[tokindex].up = longest;
			//result[tokindex].lcpup = firstfindhitlen;				
		} else {
			R = firstfindhitR;
			L = firstfindhit;
			while (R - L > 1){
				M = (L + R) >> 1;
				holdtemp = (L + M)>>1;

				if (L == M-1){
					skip = reflcpleft[M]; 
				} else{
					skip = fminf(reflcpleft[holdtemp], reflcpright[holdtemp]); //LCP(M, M')
				}

				//printf("!! Matched= R%d - L%d - M%d - LCP(L,M)%d\n", R, L, M, skip);
				if (skip >= 1){
					longest = M;
					L = M;
					//firstfindhitlen = skip;
				} else {
					R = M;
				}
			}	

			////Write results
			result[tokindex].down = longest;
			//result[tokindex].lcpdown = firstfindhitlen;
		}

	}

}

void * suffixArraySearchInit(int refbufsize, int qrysbufsize, int connectsize, int sasize) {
	hipError_t err;
	
	size_t freeMem = 0;
	size_t totalMem = 0;
	size_t allocMem = 0;

	hipMemGetInfo(&freeMem, &totalMem);  
	fprintf(stderr, "Memory avaliable: Free: %lu, Total: %lu. Start of Suffix.cu\n",freeMem/(1024*1024), totalMem/(1024*1024));

	context_t * ctx = (context_t *)malloc(sizeof(context_t));

	/* allocate the queries buffer */   
	hipMalloc((void**)&(ctx->qryset_d.qrysbuf), qrysbufsize);

	/* allocate space for the suffix array data (SA, LCP),
	   the reference itself will be allocated separately */
	hipMalloc((void**)&(ctx->ref_d.buf), refbufsize);
	hipMalloc((void**)&(ctx->ref_d.sa), sasize);
	hipMalloc((void**)&(ctx->ref_d.str), sasize);
	hipMalloc((void**)&(ctx->qryset_d.connectoffset), connectsize);
	return (void *)ctx;
}

void suffixArraySearchFinalize_One(void *handler) {

	context_t * ctx = (context_t *)handler;

	hipFree(ctx->qryset_d.qrysbuf);
	hipFree(ctx->qryset_d.result_two);	
	hipFree(ctx->qryset_d.connectoffset);
	hipFree(ctx->qryset_d.result_connect);
	hipFree(ctx->qryset_d.onegap);
	//hipFree(ctx->ref_d.buf);//lcp, rcp stuff
	//hipFree(ctx->ref_d.str);
}


void suffixArraySearchFinalize(void *handler) {

	context_t * ctx = (context_t *)handler;	

	hipFree(ctx->ref_d.sa);
	free(ctx);
}


__global__ void twoGapEnumeration(
		int* qrysoffsettok,
		int qryscount,    
		int tokenscount,
		result_t_two* qryresult,
		int totalconnect,
		int* tokindex_qryindex,
		gappy_search* searchGap,
		twogappy* twoGap,
		twoGapPattern* twogapPattern,
		unsigned int* counter,
		int distinctOneGapCount,
		gappy* oneGap,
		unsigned int oneGapEnuCount) {

	int searchPos = blockIdx.y*gridDim.x+blockIdx.x;
	if (searchPos >= distinctOneGapCount){
		return;
	}
	if(searchGap[searchPos].start_on_salist == -1||
			searchGap[searchPos].end_on_salist==-1){
		return;
	}

	int limit_symbol = MAX_rule_symbols - 1 - 1 - 
		searchGap[searchPos].qrystart_len - searchGap[searchPos].qryend_len;
	if(limit_symbol < -1){
		printf("Not possible symbol number in one gap kernel not ok! two Gap enu - limit_symbol %d; searchGap[searchPos].qrystart_len %d; searchGap[searchPos].qryend_len %d\n", limit_symbol, 
				searchGap[searchPos].qrystart_len,
				searchGap[searchPos].qryend_len);
		return;
	}		
	if(limit_symbol < 1){
		return;
	}

	unsigned int ender; //end positions on the onegap array
	if(searchPos == distinctOneGapCount - 1){
		ender = oneGapEnuCount;
		//Debug
		if(oneGapEnuCount <= searchGap[searchPos].position){
			printf("Not possible here Inside Two gap Enumeration\n");
			return;
		}
	} else {
		ender = searchGap[searchPos+1].position;
	}

	int threadx = threadIdx.x+searchGap[searchPos].position;
	int searchStart;
	int search_tokindex;

	int longest_len_end;
	int longest_end_iter = 1;
	unsigned int nextpos = 0;
	int i = 0;
	int qryindex;
	int end; //The first word of the next sentence

	while(threadx < ender){
		searchStart = oneGap[threadx].qrystart+	
			oneGap[threadx].qrystart_len+
			oneGap[threadx].gap+
			oneGap[threadx].qryend_len-1;
		search_tokindex =  searchStart+MIN_gap_size+1;

		//should minus the Min_gap_size
		if(searchStart <= tokenscount-1){
				//&& qrysoffsettok[qryscount+search_tokindex-1] != -1
				//&& qrysoffsettok[qryscount+search_tokindex] != -1){
			qryindex = tokindex_qryindex[searchStart];	
			if (qryindex != qryscount-1){
				end = qrysoffsettok[qryindex+1];
			} else {
				end = tokenscount;
			}

			while(search_tokindex < end ){//Note: && qrysoffsettok[qryscount+search_tokindex] != -1 check? - no!
				longest_len_end = qryresult[search_tokindex].longestmatch;
				longest_end_iter = 1;

				while(longest_end_iter<= limit_symbol
						&& longest_end_iter <= longest_len_end
						&& search_tokindex - oneGap[threadx].qrystart +  longest_end_iter - 1 <= MAX_rule_span_pattern){
					nextpos = atomicAdd(counter, 1);
					twoGap[nextpos].qryend_len = longest_end_iter;
					twoGap[nextpos].gap2 = search_tokindex;//Change it to unsigned int
					twoGap[nextpos].blockid =  searchPos;

					/*printf("Two Gap ENU|Pos %d - tokindex %d - qrystart_len %d - qryend_len1 %d - search_tok1 %d - qryend_len2 %d - search_tok2 %d|real %d|OneID %d\n", 		
						nextpos, oneGap[threadx].qrystart, oneGap[threadx].qrystart_len, oneGap[threadx].qryend_len, searchStart, longest_len_end, search_tokindex, qrysoffsettok[qryscount+search_tokindex], searchPos);*/
					///get the real aXb for sorting purpose.
					for(i=0; i < MAX_rule_symbols-4; i++){
						if(i < longest_end_iter){
							twogapPattern[nextpos].pattern[i] = 
								qrysoffsettok[qryscount+search_tokindex+i];							
						} else {
							twogapPattern[nextpos].pattern[i] = -2; 
						}
					}
					twogapPattern[nextpos].number = longest_end_iter;
					twogapPattern[nextpos].blockid = searchPos;
					longest_end_iter++; 			
				}			
				search_tokindex++; 
			}		
		}
		threadx += blockDim.x;
	}

}

__global__ void oneGapEnumeration(
		int* qrysoffsettok,
		int qryscount,    
		int tokenscount,
		result_t_two* qryresult,
		int totalconnect,
		int* tokindex_qryindex,
		gappy* onegap,
		gapPattern* onegapPattern,
		unsigned int* counter) {
	//max_rule_span -> 15; Maximum rule span
	//max_rule_symbols -> 5; Maximum number of symbols (terminals + nontermals) in a rule
	//min_gap_size -> 1; Minimum gap size
	//max_nonterminals -> 2; Maximum number of nonterminals in a rule
	//max_phrase_len -> 4; Maximum frequent phrase length

	int tokindex = threadIdx.x+blockIdx.y*blockDim.x;
	if(tokindex >= tokenscount-1){
		return;
	}
	int qryindex = tokindex_qryindex[tokindex];
	int end; //The first word of the next sentence

	if (qryindex != qryscount-1){
		end = qrysoffsettok[qryindex+1];
	} else {
		end = tokenscount;
	}

	if(qryindex >= qryscount 
			|| qryindex < 0 
			|| tokindex == end-1 
			|| tokindex == end-2){
		return;
	}
	int longest_len_start = qryresult[tokindex].longestmatch;
	int search_tokindex;
	int longest_len_end;
	int longest_end_iter = 1;
	int longest_start_iter = 1;
	unsigned int nextpos = 0;
	int i = 0;
	bool gappyYes = false; ///for print gappy pattern; bug fix; indicate whether it passes gaps or not
	uint8_t prePos = 0;

	/*if(qrysoffsettok[qryscount+tokindex]==-1){
		printf("Interesting MISTAKE inside one gap enu: longest match %d\n", longest_len_start);
		return;
	}*/
	
	while(longest_start_iter <= longest_len_start){
		search_tokindex = tokindex+longest_start_iter+MIN_gap_size;
		while(search_tokindex < end && search_tokindex - tokindex <= MAX_rule_span_pattern){
			if(qrysoffsettok[qryscount+search_tokindex]!=-1){
				longest_len_end = qryresult[search_tokindex].longestmatch;
				longest_end_iter = 1;

				while(longest_start_iter + 1 + longest_end_iter<= MAX_rule_symbols
						&& longest_end_iter <= longest_len_end
						&& search_tokindex - tokindex + longest_end_iter - 1<= MAX_rule_span_pattern){
					nextpos = atomicAdd(counter, 1);
					onegap[nextpos].qrystart = tokindex;
					onegap[nextpos].qrystart_len = longest_start_iter;
					onegap[nextpos].qryend_len = longest_end_iter;
					onegap[nextpos].gap = search_tokindex - tokindex - longest_start_iter;
					/*printf("One Gap ENU|Pos %d - tokindex %d - qrystart_len %d - qryend_len %d - search_to %d - gap %d\n", 
						nextpos, tokindex,longest_start_iter, longest_end_iter, search_tokindex, 
						onegap[nextpos].gap);*/
					///get the real aXb for sorting purpose.
					gappyYes = false;
					prePos = 0;
					for(i=0; i < MAX_rule_symbols; i++){
						if(i < longest_start_iter + 1 + longest_end_iter){
							if(!gappyYes){
								if(i == longest_start_iter + 1 - 1){
									onegapPattern[nextpos].pattern[i] = -1;
									gappyYes = true;
									prePos = i;
									//represents the gap
								} else {
									onegapPattern[nextpos].pattern[i] = 
										qrysoffsettok[qryscount+tokindex+i];
									/*if(qrysoffsettok[qryscount+tokindex+i]==-1){
									  printf("Wrong! Kernel one gap enu 1! -1|longest_len_end %d |longest_len_iter %d|\n", 
									  longest_len_end, longest_end_iter);
									  }*/
								}
							} else {
								onegapPattern[nextpos].pattern[i] = 
									qrysoffsettok[qryscount+search_tokindex+i-1-prePos];
								/*if(search_tokindex+i-1-prePos>search_tokindex){
									printf("Wrong!!! inside one gap enu pattern generation\n");
									return;
								}									
								if(qrysoffsettok[qryscount+search_tokindex+i-1-prePos]==-1){
								  printf("Wrong! Kernel one gap enu 2! -1|longest_len_end %d |longest_len_iter %d|\n", 
								  longest_len_end, longest_end_iter);
								  }*/
							}
						} else {
							onegapPattern[nextpos].pattern[i] = -2;
						}
					}
					onegapPattern[nextpos].number = longest_start_iter+1+longest_end_iter;
					longest_end_iter++;				
				}			
			}
			search_tokindex++; 
		}
		longest_start_iter++;
	}			
}

__global__ void zeroOneDiff(
		uint8_t* zeroOneDiffArray_d,
		gapPattern* onegapPattern,
		unsigned int counter) {
	
	int blocks = gridDim.x*blockIdx.y+blockIdx.x;
	unsigned int index = threadIdx.x+blocks*blockDim.x;
	
	bool stop = true;
	if(index < counter){
		//zeroOneDiffArray_d[index] = 0;
		if(index ==0){
			zeroOneDiffArray_d[index] = 1;
		} else {
			if (onegapPattern[index-1].number != onegapPattern[index].number){		
				zeroOneDiffArray_d[index] = 1;
			} else {
				for(int i =0; i < onegapPattern[index].number && stop; i++){
					if(onegapPattern[index-1].pattern[i] 
							!= onegapPattern[index].pattern[i]){
						zeroOneDiffArray_d[index] = 1;
						stop = false;
					}
				}		
			}
		}
	}
}

__global__ void zeroOneDiffTwoGap(
		uint8_t* zeroOneDiffArray_d,
		twoGapPattern* twoGapPattern,
		unsigned int counter) {
			
	int blocks = gridDim.x*blockIdx.y+blockIdx.x;
	unsigned int index = threadIdx.x+blocks*blockDim.x;
	
	//unsigned int index = threadIdx.x+blockIdx.y*blockDim.x;
	if(index >= counter){
		return;
	}

	if(index ==0){
		zeroOneDiffArray_d[index] = 1;
		return;
	}

	if (twoGapPattern[index-1].number != twoGapPattern[index].number){		
		zeroOneDiffArray_d[index] = 1;
		return;
	} else if (twoGapPattern[index-1].blockid != twoGapPattern[index].blockid){
		zeroOneDiffArray_d[index] = 1;
		return;
	}

	for(int i =0; i < twoGapPattern[index].number; i++){
		if(twoGapPattern[index-1].pattern[i] 
				!= twoGapPattern[index].pattern[i]){
			zeroOneDiffArray_d[index] = 1;
			return;
		}
	}

	zeroOneDiffArray_d[index] = 0;		
}


	int compareUserTotal1(const void *v1, const void *v2)
	{
		const precomp_tok* u1 = (precomp_tok*) v1;
		const precomp_tok* u2 = (precomp_tok*) v2;
		return (u1->length < u2->length);
	}


	int compareUserTotal2(const void *v1, const void *v2)
	{
		const precomp_tok* u1 = (precomp_tok*) v1;
		const precomp_tok* u2 = (precomp_tok*) v2;
		return (u1->token  > u2->token);
	}

	int compareUserTotal3(const void *v1, const void *v2)
	{
		const precompute_enu_2* u1 = (precompute_enu_2*) v1;
		const precompute_enu_2* u2 = (precompute_enu_2*) v2;
		return (u1->index  > u2->index) 
			|| ((u1->index == u2->index) && (u1->start > u2->start )) 
			|| ((u1->index == u2->index) && (u1->start == u2->start ) && (u1->length > u2->length));
	}

	void preComputation(
			int* sa_d, 
			int* str_d, 
			ref_t* ref,
			unsigned int* RLP_d,
			uint8_t* L_target_d,
			uint8_t* R_target_d){
		mytimer_t __t;
		timer_start(&__t);

		int i = 0;
		int prevtoken;
		size_t freeMem = 0;
		size_t totalMem = 0;
		size_t allocMem = 0;

		while(ref->str[ref->sa[i]] < 2){
			i++;
			prevtoken = ref->str[ref->sa[i]];
		}
		int prevTokCount = 0;
		int counter_toplist = 0;
		precomp_tok* toplist = (precomp_tok*)malloc(ref->distinctTokenCount*sizeof(precomp_tok));
		i = 0;
		for(; i< ref->toklen; i++){
			if (ref->str[ref->sa[i]] < 2){
				continue;
			}
			if(prevtoken != ref->str[ref->sa[i]]){		
				toplist[counter_toplist].end = i-1;
				toplist[counter_toplist].length = prevTokCount;
				toplist[counter_toplist].token = prevtoken;
				counter_toplist++;
				prevtoken =  ref->str[ref->sa[i]]; 	
				prevTokCount = 1;
			} else {
				prevTokCount++;
			}
		}	
		toplist[counter_toplist].end = i-1;
		toplist[counter_toplist].length = prevTokCount;
		toplist[counter_toplist].token = prevtoken;
		counter_toplist++;
		qsort(toplist, counter_toplist, sizeof(precomp_tok), compareUserTotal1);
		qsort(toplist, PRECOMPUTECOUNT, sizeof(precomp_tok), compareUserTotal2);

		//verification - top 1000 chinese words in Suffix array - looks good.
		/*for(int j=0; j< PRECOMPUTECOUNT; j++){
		  if (toplist[j].token < 2){			
		  fprintf(stderr, "NOT POSSIBLE! LESS Than 2\n");
		//continue;
		}
		struct my_struct *s;
		HASH_FIND_INT(intchar, &toplist[j].token, s );	
		if(!s){
		printf("CANNOT FIND this WORD! - IMPOSSIBLE!!  -->");
		}
		printf("%d -> LEN %d | Tok %d - %s| E %d\n", j, toplist[j].length, toplist[j].token, s->name, toplist[j].end);		
		}*/

		ref->frequentList = (int*)malloc(PRECOMPUTECOUNT*sizeof(int));
		for(int j=0; j< PRECOMPUTECOUNT; j++){
			if (toplist[j].token < 2){			
				fprintf(stderr, "NOT POSSIBLE! LESS Than 2\n");
			}
			ref->frequentList[j] = toplist[j].token;
		}

		precompute_enu* oneGapPrecomp = (precompute_enu*)malloc(PRECOMPUTECOUNT*PRECOMPUTECOUNT*sizeof(precompute_enu));
		for(int cc=0; cc<PRECOMPUTECOUNT ; cc++){		
			for(int jj=0; jj<PRECOMPUTECOUNT; jj++){
				if(toplist[jj].length >= toplist[cc].length){
					oneGapPrecomp[cc*PRECOMPUTECOUNT+jj].length = toplist[cc].length;
					oneGapPrecomp[cc*PRECOMPUTECOUNT+jj].token_a = toplist[cc].token;				
					oneGapPrecomp[cc*PRECOMPUTECOUNT+jj].token_b = toplist[jj].token;
					oneGapPrecomp[cc*PRECOMPUTECOUNT+jj].start = toplist[cc].end - toplist[cc].length + 1;
					oneGapPrecomp[cc*PRECOMPUTECOUNT+jj].reverse = true;
				} else {
					oneGapPrecomp[cc*PRECOMPUTECOUNT+jj].length = toplist[jj].length;
					oneGapPrecomp[cc*PRECOMPUTECOUNT+jj].token_a = toplist[cc].token;				
					oneGapPrecomp[cc*PRECOMPUTECOUNT+jj].token_b = toplist[jj].token;
					oneGapPrecomp[cc*PRECOMPUTECOUNT+jj].start = toplist[jj].end - toplist[jj].length + 1;
					oneGapPrecomp[cc*PRECOMPUTECOUNT+jj].reverse = false;
				}
			}
		}	

		//Verification on OneGapPrecomp
		/*for(int icao = 0; icao<PRECOMPUTECOUNT*PRECOMPUTECOUNT; icao++){
		  int cc = icao / PRECOMPUTECOUNT;
		  int jj = icao % PRECOMPUTECOUNT;

		  printf("%d - ID A %d; ID B %d - Len A %d; Len B %d - End len %d \n",
		  icao,
		  oneGapPrecomp[icao].token_a,
		  oneGapPrecomp[icao].token_b,
		  toplist[cc].length,
		  toplist[jj].length,
		  oneGapPrecomp[icao].length);
		  }*/

		////CUDA Starts
		int* featureMissingCount_d;
    hipMalloc((void**)&(featureMissingCount_d), 
                  PRECOMPUTECOUNT*PRECOMPUTECOUNT*sizeof(int));
		precompute_enu* oneGapPrecomp_d;
		hipMalloc((void**)&(oneGapPrecomp_d), PRECOMPUTECOUNT*PRECOMPUTECOUNT*sizeof(precompute_enu));
		hipMemcpy(oneGapPrecomp_d, oneGapPrecomp, PRECOMPUTECOUNT*PRECOMPUTECOUNT*sizeof(precompute_enu), hipMemcpyHostToDevice);

		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable: Free: %lu, Total: %lu\n",freeMem/(1024*1024), totalMem/(1024*1024));

		unsigned int count = 0, *count_d;
		hipMalloc((void**)&count_d, sizeof(unsigned int));
		hipMemcpy(count_d, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

		precompute_enu_2* oneGapPrecomp_result_d;
		//precompute_enu_2* oneGapPrecomp_result;
		hipMalloc((void**)&(oneGapPrecomp_result_d), sizeof(precompute_enu_2)*ONEGAP_PRECOMPUT_PREALLOCATION);

		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable: ONEGAP - Free: %lu, Total: %lu\n",freeMem/(1024*1024), totalMem/(1024*1024));

#define THREADS_PER_BLOCK 512 
		dim3 block3(THREADS_PER_BLOCK, 1);
		//dim3 grid3(20000, 1);//Tune this and see the difference!
		dim3 grid3(1000, (PRECOMPUTECOUNT*PRECOMPUTECOUNT+1000-1)/1000);

		assert(str_d!=NULL && sa_d != NULL && oneGapPrecomp_result_d != NULL
			&& RLP_d!=NULL && L_target_d!=NULL && R_target_d!=NULL);
		precomp<<<grid3, block3>>>(
			oneGapPrecomp_d, 
			str_d, 
			sa_d, 
			oneGapPrecomp_result_d, 
			count_d,
			RLP_d,
			L_target_d,
			R_target_d,
			featureMissingCount_d);
		hipDeviceSynchronize();

		hipError_t error = hipGetLastError();
		if(error != hipSuccess)
		{
			fprintf(stderr, "CUDA error second After Precomputation Kernel: %s \n", hipGetErrorString(error));
			exit(-1);
		}	
		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable: Free: %lu, Total: %lu\n",freeMem/(1024*1024), totalMem/(1024*1024));

		hipMemcpy(&count, count_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
		fprintf(stderr, "Found %u pairs!!\n", count);

		precompute_enu_2* precomp_onegap_large;
		hipHostMalloc((void**)&(precomp_onegap_large), count*sizeof(precompute_enu_2));
		hipMemcpy(precomp_onegap_large, oneGapPrecomp_result_d, count*sizeof(precompute_enu_2), hipMemcpyDeviceToHost);

		ref->featureMissingCount = (int*)malloc(sizeof(int)*PRECOMPUTECOUNT*PRECOMPUTECOUNT);
    hipMemcpy(ref->featureMissingCount, featureMissingCount_d, 
                        sizeof(int)*PRECOMPUTECOUNT*PRECOMPUTECOUNT, hipMemcpyDeviceToHost);
		///Process results
		hipFree(featureMissingCount_d);
		hipFree(oneGapPrecomp_result_d);	
		hipFree(oneGapPrecomp_d);	
		free(oneGapPrecomp);
		free(toplist);

		qsort(precomp_onegap_large, count, sizeof(precompute_enu_2), compareUserTotal3);
		hipHostMalloc((void **)&(ref->precomp_index), PRECOMPUTECOUNT*PRECOMPUTECOUNT*sizeof(precomp_st_end));
		unsigned int prevC=0;
		bool firsttime = true;
		for(unsigned int ic = 0; ic<PRECOMPUTECOUNT*PRECOMPUTECOUNT; ic++ ){
			firsttime = true;
			ref->precomp_index[ic].start = 1;
			ref->precomp_index[ic].end = 0;
			while(precomp_onegap_large[prevC].index == ic){
				if(firsttime){
					ref->precomp_index[ic].start = prevC;	
					firsttime = false;
				}
				ref->precomp_index[ic].end = prevC;
				prevC++;
			}
		}

		/*for(unsigned int ic = 0; ic<PRECOMPUTECOUNT*PRECOMPUTECOUNT; ic++ ){
		  if(ref->precomp_index[ic].start == -1){
		  continue;
		  }
		  printf("%d -> s %d | e%d\n", ic, ref->precomp_index[ic].start, ref->prec);
		  }*/

		hipHostMalloc((void**)&(ref->precomp_onegap), count*sizeof(precompute_enu_3));
		for(unsigned int yoshi = 0; yoshi < count; yoshi++){
			if(precomp_onegap_large[yoshi].start+precomp_onegap_large[yoshi].length > 
				ref->toklen || precomp_onegap_large[yoshi].length <= 0 
				|| precomp_onegap_large[yoshi].length+1 > MAX_rule_span){
				printf("Not possible inside precomputation!|index %d|end %d\n", 
				yoshi, precomp_onegap_large[yoshi].start+precomp_onegap_large[yoshi].length);
			}
			ref->precomp_onegap[yoshi].start = precomp_onegap_large[yoshi].start;
			ref->precomp_onegap[yoshi].length = precomp_onegap_large[yoshi].length;
		}
		hipHostFree(precomp_onegap_large);
		ref->precomp_count = count;
		timer_stop(&__t);	
		fprintf(stderr, "-> Precmputation time: %f\n", timer_elapsed(&__t)/1000);	
	}

void suffixArraySearch(
		void *handler,
		ref_set_t *refset, 
		qry_set_t *qryset, 
		int minmatch,
		timing_t *timing,
		int twoway) {

	mytimer_t __t;
	hipError_t err;
	context_t * ctx = (context_t *)handler;

	ref_t *ref_d	= &ctx->ref_d;
	ref_t_target* ref_target_d = &ctx->ref_target_d;	

	qry_set_t *qryset_d = &ctx->qryset_d;
	size_t freeMem = 0;
	size_t totalMem = 0;
	size_t allocMem = 0;

	/*- - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - - -*/
	qryset_d->qrysoffsettok = (int *)qryset_d->qrysbuf; 
	qryset_d->qryscount  = qryset->qryscount;
	qryset_d->totaltokens =qryset->totaltokens; 

	timer_start(&__t);
	/* move the query set to the device */
	hipMemcpy(qryset_d->qrysoffsettok, qryset->qrysbuf, qryset->qrysbufsize, hipMemcpyHostToDevice);
	timing->qrysin += timer_stop(&__t);

	std::cout << "Thrust v" << THRUST_MAJOR_VERSION << "." << THRUST_MINOR_VERSION << std::endl;

#define THREADS_PER_BLOCK1 128 
#define Qry_Per_BLOCK 4 

	dim3  block2(THREADS_PER_BLOCK1, Qry_Per_BLOCK);//TBD!!!!
	dim3  grid2(2 ,(qryset->qryscount+block2.y-1)/block2.y);

	int refindex;
	for (refindex = 0;	refindex < refset->count;	refindex++) {
		ref_t * ref_h = &refset->refs[refindex];
		ref_t_target* ref_target_h = &refset->refs_target[refindex];	
		
		//ref_d->str 	  = (int *)ref_d->buf;
		//ref_d->sa      = (int *)ref_d->buf + (ref_h->sa - ref_h->str);
		ref_d->lcp	  = (int *)(ref_d->buf + (ref_h->lcp - ref_h->buf));
		ref_d->rank   = (int *)(ref_d->buf + (ref_h->rank - ref_h->buf));
		ref_d->lcpleft = (int *)(ref_d->buf + (ref_h->lcpleft - ref_h->buf));
		ref_d->lcpright = (int *)(ref_d->buf + (ref_h->lcpright - ref_h->buf));
		ref_d->toklen = ref_h->toklen;

		int datasize = ref_d->toklen * sizeof(int)*2;
		// Copy the sub reference to the device
		timer_start(&__t);
		hipMemcpy(ref_d->sa, (void *)ref_h->sa, ref_h->toklen*sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(ref_d->str, (void *)ref_h->str, ref_h->toklen*sizeof(int), hipMemcpyHostToDevice);		
		hipMemcpy(ref_d->buf, (void *)ref_h->buf, datasize, hipMemcpyHostToDevice);

		hipMalloc((void**)&(qryset_d->result_two), qryset->resbufsize);
		timing->refsin += timer_stop(&__t);

		/////////////////////////////////////////////
		///Precompute First! - One time offline!/////
		/////////////////////////////////////////////
		hipMalloc((void**)&(ref_d->RLP), ref_h->toklen*sizeof(unsigned int));
		hipMalloc((void**)&(ref_target_d->L_tar), ref_target_h->toklen*sizeof(uint8_t));
		hipMalloc((void**)&(ref_target_d->R_tar), ref_target_h->toklen*sizeof(uint8_t));
				
		hipMemcpy(ref_target_d->R_tar, ref_target_h->R_tar, ref_target_h->toklen*sizeof(uint8_t), hipMemcpyHostToDevice);
		hipMemcpy(ref_target_d->L_tar, ref_target_h->L_tar, ref_target_h->toklen*sizeof(uint8_t), hipMemcpyHostToDevice);
		hipMemcpy(ref_d->RLP, ref_h->RLP, ref_h->toklen*sizeof(unsigned int), hipMemcpyHostToDevice);
		
		preComputation(ref_d->sa, 
							ref_d->str, 
							ref_h, 
							ref_d->RLP,
							ref_target_d->L_tar,
							ref_target_d->R_tar);
		hipFree(ref_d->RLP);
		hipFree(ref_target_d->L_tar);
		hipFree(ref_target_d->R_tar);
		
		////////////////////////////////////////////
		/////////////////Suffix Array lookUp////////
		////////////////////////////////////////////
		timer_start(&__t);
		
		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable pass1: Free: %lu, Total: %lu\n",freeMem/(1024*1024), totalMem/(1024*1024));
		suffixArrayFindLwRwKernelTwoWayTDI<<<grid2, block2>>>(
				ref_d->str,
				ref_d->sa,
				ref_d->lcpleft,
				ref_d->lcpright,
				ref_d->toklen,
				qryset_d->qrysoffsettok,
				qryset->qryscount,
				qryset->totaltokens,
				qryset_d->result_two);

		hipDeviceSynchronize();

		timing->kernel += timer_stop(&__t);
		hipError_t error = hipGetLastError();
		fprintf(stderr, "Kernel 1 time: %f\n", timer_elapsed(&__t)/1000);
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit if any
			printf("CUDA error second After kernel 1: %s \n", hipGetErrorString(error));
			exit(0);
		}
		// Copy the results from the device
		timer_start(&__t);
		hipMemcpy(qryset->result_two,qryset_d->result_two,qryset->resbufsize, hipMemcpyDeviceToHost);
		
	error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit if any
			printf("CUDA error second IN Between Memcpy: %s \n", hipGetErrorString(error));
			exit(0);
		}
		int ii = 0;
		qryset->totalconnect = 0;
		for(;ii< qryset->totaltokens;ii++){
			//fprintf(stderr, "%d -> qryresults longest match %d\n", ii, qryset->result_two[ii].longestmatch);
			if (qryset->result_two[ii].longestmatch-1 > 0){
				qryset->connectoffset[ii] = qryset->totalconnect;
				qryset->totalconnect += qryset->result_two[ii].longestmatch-1;
			}else {
				qryset->connectoffset[ii] = -1;
			}
		}

		timing->resout += timer_stop(&__t);
		timer_start(&__t);
#define THREADS_PER_BLOCK1 128
#define LONGEST_PER_BLOCK 6 
		dim3  block(THREADS_PER_BLOCK1, 1);
		dim3  grid((qryset->totaltokens+block.x -1)/block.x ,LONGEST_PER_BLOCK);
		//printf("qryset->totalconnect %d\n", qryset->totalconnect);
		hipMalloc((void**)&(qryset_d->result_connect), qryset->totalconnect*sizeof(result_t));
		hipMemcpy(qryset_d->connectoffset, qryset->connectoffset, qryset->totaltokens*sizeof(int), hipMemcpyHostToDevice);
		timing->qrysin += timer_stop(&__t); 

		timer_start(&__t);
		
		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable pass2: Free: %lu, Total: %lu\n",freeMem/(1024*1024), totalMem/(1024*1024));
		suffixArrayFindConnectionTwoWayTDI<<<grid, block>>>(
				ref_d->str,
				ref_d->sa,
				/*ref_d->lcp,*/						
				ref_d->lcpleft,
				ref_d->lcpright,
				ref_d->toklen,
				qryset_d->qrysoffsettok,
				qryset->qryscount,
				qryset->totaltokens,
				qryset_d->result_two,
				qryset_d->connectoffset,
				qryset->totalconnect,
				qryset_d->result_connect);

		hipDeviceSynchronize();					
		timing->kernel += timer_stop(&__t);
		timer_start(&__t);
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			// print the CUDA error message and exit if any
			printf("CUDA error second Second Kernel: %s \n", hipGetErrorString(error));
			//exit(-1);
		}

		hipHostMalloc((void **)&(qryset->result_connect), qryset->totalconnect*sizeof(result_t));
		hipMemcpy(qryset->result_connect,qryset_d->result_connect, qryset->totalconnect*sizeof(result_t), hipMemcpyDeviceToHost);

		timing->resout += timer_stop(&__t);
		fprintf(stderr, "Result out time: %f second\n", timer_elapsed(&__t)/1000);      
		fprintf(stderr, "Total Continous Phrase Kernel time: %f second\n", timing->kernel/1000);     

		//////////////////////////////////////////////////////////////////
		////////////////////////////////////////Gappy Phrase Lookup///////
		//////////////////////////////////////////////////////////////////
		hipFree(ref_d->buf);

		//////////////////////////////////////////////////One gap enu!
#define THREADS_PER_BLOCK_GAP 32
		dim3  block_gap_enu(THREADS_PER_BLOCK_GAP, 1);
		dim3  grid_one_gap_enu(1, (qryset->totaltokens+THREADS_PER_BLOCK_GAP-1)/THREADS_PER_BLOCK_GAP);

		hipMalloc((void**)&(qryset_d->tokindex_qryindex), qryset->totaltokens*sizeof(int));
		hipMemcpy(qryset_d->tokindex_qryindex, (void *)qryset->tokindex_qryindex, qryset->totaltokens*sizeof(int), hipMemcpyHostToDevice);

		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable: Free: %lu, Total: %lu\n",freeMem/(1024*1024), totalMem/(1024*1024));

		gappy* onegap;
		hipMalloc((void**)&(onegap), sizeof(gappy)*ONEGAP_ENU_PREALLOCATION);
		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable for one gap without pattern: Free: %lu, Total: %lu\n",freeMem/(1024*1024), totalMem/(1024*1024));

		gapPattern* onegapPattern;
		hipMalloc((void**)&(onegapPattern), sizeof(gapPattern)*ONEGAP_ENU_PREALLOCATION);
		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable for one gap with pattern: Free: %lu, Total: %lu\n",freeMem/(1024*1024), totalMem/(1024*1024));

		unsigned int count = 0, *count_d;
		hipMalloc((void**)&count_d, sizeof(unsigned int));
		hipMemcpy(count_d, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

		error = hipGetLastError();
		if(error != hipSuccess)
		{
			fprintf(stderr, "CUDA error second Before kernel One Gap First - Phrase Lookup: %s \n", hipGetErrorString(error));
			exit(-1);
		}	
		
		//Debug
		//fprintf(stderr, "Tokenscount %d | qryresult[tokenscount-1] %d\n", qryset->totaltokens, qryset->result_two[qryset->totaltokens-2].longestmatch);	
		timer_start(&__t);			
		assert(qryset_d->tokindex_qryindex!=NULL 
				&& onegap!=NULL
				&& qryset_d->qrysoffsettok != NULL);

		oneGapEnumeration<<<grid_one_gap_enu, block_gap_enu>>>(
				qryset_d->qrysoffsettok,
				qryset->qryscount,
				qryset->totaltokens,
				qryset_d->result_two,//qryresult
				qryset->totalconnect,
				qryset_d->tokindex_qryindex,
				onegap,
				onegapPattern,
				count_d);

		hipDeviceSynchronize();		
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			fprintf(stderr, "CUDA error second After kernel One Gap Enumer: %s \n", hipGetErrorString(error));
			exit(-1);
		}	
		hipMemcpy(&count, count_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
		timer_stop(&__t);
		fprintf(stderr, "Found %u pairs for one gap enumeration!!\n", count);
		fprintf(stderr, "-> One Gap Enumeration time: %f\n", timer_elapsed(&__t)/1000);      

		/////////////Sorts these gap aXb based on gappy patterns
		timer_start(&__t);		
		thrust::device_ptr<gappytyp> onegap_thrust = thrust::device_pointer_cast(onegap);
		thrust::device_ptr<gapPattern1> onegapPattern_thrust = thrust::device_pointer_cast(onegapPattern);
		thrust::device_ptr<gapPattern1> onegapPattern_thrust_end = thrust::device_pointer_cast(onegapPattern+count);

		//sort by key! with thrust
		thrust::sort_by_key(onegapPattern_thrust, onegapPattern_thrust_end, onegap_thrust, oneGapEnumerationCompare());

		hipMemGetInfo(&freeMem, &totalMem);  		
		fprintf(stderr, "Memory avaliable: Free: %lu, Total: %lu - Duplication Memory - After Sort\n",freeMem/(1024*1024), totalMem/(1024*1024));

		//Debugiing!! - Show unsorted onegap pattern
		/*gapPattern* debugUnsorterPattern;
		hipHostMalloc((void**)&(debugUnsorterPattern), count*sizeof(gapPattern));
		hipMemcpy(debugUnsorterPattern, onegapPattern, count*sizeof(gapPattern), hipMemcpyDeviceToHost);
		for(int k=0;k < count; k++ ){
		///Debuging see sorted one gap pattern 
		printf("Sorted one gap| No. %d --> ", k);
		for(int j=0; j< MAX_rule_symbols; j++){
		printf("%d ", debugUnsorterPattern[k].pattern[j]);					
		}
		printf(" |\n");
		///Debugging end				
		}*/
		///
		
		///Get 1-0 bit array to identify the differences for later sequential scan process
#define THREADS_PER_BLOCK_DIFF 256
		dim3 block_gap_diff(THREADS_PER_BLOCK_DIFF, 1);
		dim3 grid_one_gap_diff(10, (count+THREADS_PER_BLOCK_DIFF*10-1)/(10*THREADS_PER_BLOCK_DIFF));

		uint8_t* zeroOneDiffArray_d;
		hipMalloc((void**)&(zeroOneDiffArray_d), sizeof(uint8_t)*(count+1));
		hipMemset(zeroOneDiffArray_d, 0, sizeof(uint8_t)*(count+1));

		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable after zero one array: Free: %lu, Total: %lu\n",freeMem/(1024*1024), totalMem/(1024*1024));
		
		printf("ZeroOoneDiff -> %d %d\n", grid_one_gap_diff.x, grid_one_gap_diff.y);
		zeroOneDiff<<<grid_one_gap_diff, block_gap_diff>>>(
				zeroOneDiffArray_d,
				onegapPattern,
				count);

		hipDeviceSynchronize();		
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			fprintf(stderr, "CUDA error second After kernel - Zero One Diff: %s \n", hipGetErrorString(error));
			exit(-1);
		}	

		uint8_t* zeroOneDiffArray;
		hipHostMalloc((void **)&(zeroOneDiffArray), count*sizeof(uint8_t));
		hipMemcpy(zeroOneDiffArray,zeroOneDiffArray_d, count*sizeof(uint8_t), hipMemcpyDeviceToHost);

		////////////////Now get sequential scan done			
		qryset->onegapcount_enu = count;
		hipHostMalloc((void**)&(qryset->onegap), qryset->onegapcount_enu*sizeof(gappy));
		hipMemcpy(qryset->onegap, onegap, qryset->onegapcount_enu*sizeof(gappy), hipMemcpyDeviceToHost);

		gappy_search* oneGapSearch, *oneGapSearch_d;
		hipHostMalloc((void **)&(oneGapSearch), count*sizeof(gappy_search));

		std::vector<std::vector<unsigned int> > oneGapQueryWithID(qryset->qryscount);
		std::map<unsigned int, bool> checkDup;
		std::map<unsigned int, bool>::iterator iter_check;

		/////////Get the sorted stuff out and see gappy phrase		
		hipHostMalloc((void**)&(qryset->onegapPattern), qryset->onegapcount_enu*sizeof(gapPattern));
		hipMemcpy(qryset->onegapPattern, onegapPattern, qryset->onegapcount_enu*sizeof(gapPattern), hipMemcpyDeviceToHost);
		timer_stop(&__t);
		fprintf(stderr, "-> One Gap Enumeration Sorting time: %f\n", timer_elapsed(&__t)/1000);      
		
		//scan
		timer_start(&__t);
		unsigned int distinctOneGapCount =0;
		int initialPos = 0;
		for(int i = 0; i< count; i++){
			///Debuging see sorted one gap pattern 
			/*printf("No. %d --> ", i);
			  for(int j=0; j< MAX_rule_symbols; j++){
			  printf("%d ", qryset->onegapPattern[i].pattern[j]);					
			  }
			  printf(" ||| Diff %d\n",zeroOneDiffArray[i]);*/
			///Debugging end

			if(zeroOneDiffArray[i]==1){
				checkDup.clear();
				initialPos = i;
				oneGapSearch[distinctOneGapCount].position = i;
				oneGapSearch[distinctOneGapCount].gap = qryset->onegap[i].gap;
				oneGapSearch[distinctOneGapCount].qryend_len = qryset->onegap[i].qryend_len;
				oneGapSearch[distinctOneGapCount].qrystart = qryset->onegap[i].qrystart;
				oneGapSearch[distinctOneGapCount].qrystart_len = qryset->onegap[i].qrystart_len;
				oneGapSearch[distinctOneGapCount].end_on_salist = -1;
				oneGapSearch[distinctOneGapCount].start_on_salist = -1;
				
				if(qryset->onegap[i].gap<=0 || qryset->onegap[i].qrystart <0){
					printf("No. %d --> qrystart %d - searchInd %d - gap %d - qrystartlen %d\n", distinctOneGapCount,
						oneGapSearch[distinctOneGapCount].qrystart, 
						oneGapSearch[distinctOneGapCount].qrystart+oneGapSearch[distinctOneGapCount].gap + oneGapSearch[distinctOneGapCount].qrystart_len,
						oneGapSearch[distinctOneGapCount].gap,
						oneGapSearch[distinctOneGapCount].qrystart_len);
				}
				distinctOneGapCount ++;
				///Debuging see sorted one gap pattern 
				/* printf("aXb -> REAL SORTED ID %d: ", distinctOneGapCount-1);
				  for(int j=0; j< MAX_rule_symbols; j++){
				  printf("%d ", qryset->onegapPattern[i].pattern[j]);					
				  }
				  printf("||| qrystart_len %d | qryend_len %d\n", qryset->onegap[i].qrystart_len, qryset->onegap[i].qryend_len);	
				*/  
				///Debugging end
			} 
			int qid = qryset->tokindex_qryindex[qryset->onegap[i].qrystart];
			if(qid > qryset->qryscount){
				printf("Not possible, Qry count exceeded!\n");
				return;
			}
			///remove duplications
			iter_check = checkDup.find(qid);
			if(iter_check == checkDup.end()){
				checkDup.insert(std::make_pair(qid, false));
				oneGapQueryWithID[qid].push_back(distinctOneGapCount-1); //all to the 
				// uniform id, innitial ID/pos
			} 
		}		
		fprintf(stderr, "Distinct one gap pattern: %d\n", distinctOneGapCount);      
		timer_stop(&__t);
                fprintf(stderr, "-> One Gap Enumeration CPU Processing time: %f\n", timer_elapsed(&__t)/1000);

		hipFree(onegap);
		hipFree(onegapPattern);
		hipFree(zeroOneDiffArray_d);
		hipHostFree(zeroOneDiffArray);
		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable after one gap enumeration finish: Free: %lu, Total: %lu - After One Gap\n",freeMem/(1024*1024), totalMem/(1024*1024));

		////////////Now Look For distinct one gappy phrases in Suffix Array
		count = 0;
		hipMemcpy(count_d, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

		hipMalloc((void**)&(oneGapSearch_d), sizeof(gappy_search)*distinctOneGapCount);
		hipMemcpy(oneGapSearch_d, oneGapSearch, distinctOneGapCount*sizeof(gappy_search), 
				hipMemcpyHostToDevice);

		oneGapOnSA* oneGapSA, *oneGapSA_d;		
		hipMalloc((void**)&(oneGapSA_d), sizeof(oneGapOnSA)*(ONEGAP_PREALLOCATION));

		hipMalloc((void**)&(ref_d->precomp_index), PRECOMPUTECOUNT*PRECOMPUTECOUNT*sizeof(precomp_st_end));
		hipMemcpy(ref_d->precomp_index, ref_h->precomp_index, PRECOMPUTECOUNT*PRECOMPUTECOUNT*sizeof(precomp_st_end), hipMemcpyHostToDevice);

		hipMalloc((void**)&(ref_d->frequentList), PRECOMPUTECOUNT*sizeof(int));
		hipMemcpy(ref_d->frequentList, ref_h->frequentList, PRECOMPUTECOUNT*sizeof(int), hipMemcpyHostToDevice);

		hipMalloc((void**)&(ref_d->precomp_onegap), ref_h->precomp_count*sizeof(precompute_enu_3));
		hipMemcpy(ref_d->precomp_onegap, ref_h->precomp_onegap, ref_h->precomp_count*sizeof(precompute_enu_3), hipMemcpyHostToDevice);

		/////RLP Array memory allocation
		hipMalloc((void**)&(ref_d->RLP), ref_h->toklen*sizeof(unsigned int));
		hipMalloc((void**)&(ref_target_d->L_tar), ref_target_h->toklen*sizeof(uint8_t));
		hipMalloc((void**)&(ref_target_d->R_tar), ref_target_h->toklen*sizeof(uint8_t));
					
		hipMemcpy(ref_target_d->R_tar, ref_target_h->R_tar, ref_target_h->toklen*sizeof(uint8_t), hipMemcpyHostToDevice);
		hipMemcpy(ref_target_d->L_tar, ref_target_h->L_tar, ref_target_h->toklen*sizeof(uint8_t), hipMemcpyHostToDevice);
		hipMemcpy(ref_d->RLP, ref_h->RLP, ref_h->toklen*sizeof(unsigned int), hipMemcpyHostToDevice);
			
		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable right before one gap lookup kernel: Free: %lu, Total: %lu - After One Gap\n",freeMem/(1024*1024), totalMem/(1024*1024));

		assert(ref_d->sa 
				&& ref_d->str
				&& ref_d->toklen
				&& qryset_d->qrysoffsettok
				&& qryset_d->result_two
				&& qryset_d->connectoffset
				&& qryset_d->result_connect
				&& qryset_d->tokindex_qryindex
				&& oneGapSA_d
				&& ref_d->frequentList
				&& ref_d->precomp_index
				&& ref_d->precomp_onegap
				&& oneGapSearch_d);

#define THREADS_PER_BLOCK_SA 128
		dim3 block_gap_onegap_sa(THREADS_PER_BLOCK_SA, 1);
		dim3 grid_one_gap_sa(500, (distinctOneGapCount+500-1)/500);

		timer_start(&__t);	
		oneGapLookUpSA<<<grid_one_gap_sa, block_gap_onegap_sa>>>(
				ref_d->str,
				ref_d->sa,				
				ref_d->toklen,
				qryset_d->qrysoffsettok,
				qryset->qryscount,
				qryset->totaltokens,
				qryset_d->result_two,//qryresult
				qryset_d->connectoffset,
				qryset->totalconnect,
				qryset_d->result_connect,
				qryset_d->tokindex_qryindex,
				oneGapSA_d,
				ref_h->precomp_count,
				ref_d->frequentList,
				ref_d->precomp_index,
				ref_d->precomp_onegap,
				count_d,
				oneGapSearch_d,
				distinctOneGapCount,
				ref_d->RLP,
				ref_target_d->L_tar,
				ref_target_d->R_tar);

		hipDeviceSynchronize();		
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			fprintf(stderr, "CUDA error second After kernel - oneGapLookUpSA: %s \n", hipGetErrorString(error));
			exit(-1);
		}	

		unsigned int countOneGapSA = 0;
		hipMemcpy(&countOneGapSA, count_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
		
		fprintf(stderr, "Found %u for one gap on SA!\n", countOneGapSA);

		timer_stop(&__t);
                fprintf(stderr, "-> One Gap Look up on SA Kernel time: %f\n", timer_elapsed(&__t)/1000);
		
		hipFree(ref_d->sa);
		hipFree(oneGapSearch_d);
		hipFree(ref_d->RLP);
		hipFree(ref_target_d->L_tar);
		hipFree(ref_target_d->R_tar);
		
		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable before thrust one gap sort: Free: %lu, Total: %lu - After One Gap\n",freeMem/(1024*1024), totalMem/(1024*1024));

		timer_start(&__t);	
		thrust::device_ptr<gappytyp_sa> onegapSA_thrust = thrust::device_pointer_cast(oneGapSA_d);
		thrust::device_ptr<gappytyp_sa> onegapSA_thrust_end = thrust::device_pointer_cast(oneGapSA_d+countOneGapSA);
		assert(onegapSA_thrust+countOneGapSA == onegapSA_thrust_end);	
		//sort with thrust based on position attribute - ID
		thrust::sort(onegapSA_thrust, onegapSA_thrust_end, oneGapSACompare());

	  	timer_stop(&__t);
                fprintf(stderr, "-> One Gap on SA Sorting Thrust time: %f\n", timer_elapsed(&__t)/1000);
	
		hipHostMalloc((void**)&(oneGapSA), countOneGapSA*sizeof(oneGapOnSA));
		hipMemcpy(oneGapSA, oneGapSA_d, countOneGapSA*sizeof(oneGapOnSA), hipMemcpyDeviceToHost);

		///Debugging
		/*for(int cc=0;cc<countOneGapSA;cc++){
		  printf("%d->%u %u %d\n", cc, oneGapSA[cc].position, oneGapSA[cc].str_position, oneGapSA[cc].length); //on str array);
		  }			*/
		////Debugging end

		hipFree(oneGapSA_d);			

	 	timer_start(&__t);	
		//Get the start_on_salist/end_on_salist done
		for(int i =0; i<countOneGapSA; i++ ){
			if (oneGapSA[i].position >= distinctOneGapCount){
				printf("Counts not POSSIBLE! Wrong in sequential - postion %d, distinctC %d\n", oneGapSA[i].position, distinctOneGapCount);
				return;
			}
			/*if (oneGapSA[i].position == 548){
				printf("oneGapSA[i].str %d length %d|pattern: ", oneGapSA[i].str_position, oneGapSA[i].length);
				for(int iccc = oneGapSA[i].str_position; iccc <= oneGapSA[i].str_position+oneGapSA[i].length; iccc++){
					printf("%d ", ref_h->str[iccc]);
				}
				printf("\n");
			}*/
			if (oneGapSearch[oneGapSA[i].position].start_on_salist == -1){
				oneGapSearch[oneGapSA[i].position].start_on_salist = i;
			} 

			if (oneGapSearch[oneGapSA[i].position].end_on_salist == -1){
				oneGapSearch[oneGapSA[i].position].end_on_salist = i;
			} else if (oneGapSearch[oneGapSA[i].position].end_on_salist < i){
				oneGapSearch[oneGapSA[i].position].end_on_salist = i;
			}				
		}
		timer_stop(&__t);
                fprintf(stderr, "-> One Gap on SA Processing time: %f\n", timer_elapsed(&__t)/1000);

		///Debugging
		
		/*for(int i =0; i<distinctOneGapCount;i++){
		   printf("ONEGAP|qrystart %d, gap %d||one gap ID %d||First POS %d||start %d, end %d||startlen %d, endlen %d\n", 
		   oneGapSearch[i].qrystart, oneGapSearch[i].gap, i, oneGapSearch[i].position,
		   oneGapSearch[i].start_on_salist, oneGapSearch[i].end_on_salist,
		   oneGapSearch[i].qrystart_len, oneGapSearch[i].qryend_len);
		}*/
		//Book keeping
		hipHostMalloc((void**)&(qryset->oneGapSearch), sizeof(gappy_search)*distinctOneGapCount);
		memcpy(qryset->oneGapSearch, oneGapSearch, sizeof(gappy_search)*distinctOneGapCount);

		//Book keeping
		hipHostMalloc((void**)&(qryset->oneGapSA), sizeof(oneGapOnSA)*countOneGapSA);
		memcpy(qryset->oneGapSA, oneGapSA, countOneGapSA*sizeof(oneGapOnSA));

		qryset->distinctOneGapCount = distinctOneGapCount;
		qryset->countOneGapSA = countOneGapSA;
		qryset->oneGapQueryWithID = oneGapQueryWithID;
		fprintf(stderr, "After one gap look up: distinctOneGapCount %d; countOneGapSA %d\n", distinctOneGapCount, countOneGapSA);
		/////////////////////////////////
		/////////////////////////////////
		/////////////////////////////////
		/////////////////////////////////
		////Two and more gaps!!//////////	

		hipMalloc((void**)&(oneGapSearch_d), sizeof(gappy_search)*distinctOneGapCount);
		hipMemcpy(oneGapSearch_d, oneGapSearch, distinctOneGapCount*sizeof(gappy_search), 
				hipMemcpyHostToDevice);

#define THREADS_PER_BLOCK_GAP_TWO 64
		dim3  block_two_gap_enu(THREADS_PER_BLOCK_GAP_TWO, 1);
		dim3  grid_two_gap_enu(LINEARBLOCK, (distinctOneGapCount+LINEARBLOCK-1)/LINEARBLOCK);

		hipMalloc((void**)&(qryset_d->onegap), sizeof(gappy)*qryset->onegapcount_enu);
		hipMemcpy(qryset_d->onegap, qryset->onegap, qryset->onegapcount_enu*sizeof(gappy), 
				hipMemcpyHostToDevice);

		twogappy* twoGap_d;
		hipMalloc((void**)&(twoGap_d), sizeof(twogappy)*TWOGAP_ENU_PREALLOCATION);
		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable for two gap without pattern: Free: %lu, Total: %lu\n",freeMem/(1024*1024), totalMem/(1024*1024));

		twoGapPattern* twogapPattern_d;
		hipMalloc((void**)&(twogapPattern_d), sizeof(twoGapPattern)*TWOGAP_ENU_PREALLOCATION);
		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable for two gap with pattern: Free: %lu, Total: %lu\n",freeMem/(1024*1024), totalMem/(1024*1024));

		count = 0;
		hipMemcpy(count_d, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

		assert(qryset_d->tokindex_qryindex!=NULL 
				&& oneGapSearch_d!=NULL
				&& qryset_d->qrysoffsettok != NULL
				&& twoGap_d != NULL
				&& twogapPattern_d != NULL
				&& qryset_d->qrysoffsettok != NULL
				&& qryset_d->result_two != NULL
				&& qryset_d->onegap != NULL);
		
		timer_start(&__t);
		twoGapEnumeration<<<grid_two_gap_enu, block_two_gap_enu>>>(
				qryset_d->qrysoffsettok,
				qryset->qryscount,
				qryset->totaltokens,
				qryset_d->result_two,//qryresult
				qryset->totalconnect,
				qryset_d->tokindex_qryindex,
				oneGapSearch_d,
				twoGap_d,
				twogapPattern_d,
				count_d,
				distinctOneGapCount,
				qryset_d->onegap,
				qryset->onegapcount_enu);

		hipDeviceSynchronize();		
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			fprintf(stderr, "CUDA error second After kernel Two Gap Enumer: %s \n", hipGetErrorString(error));
			exit(-1);
		}	

		unsigned int countTwoGapEnu = 0;
		hipMemcpy(&countTwoGapEnu, count_d, sizeof(unsigned int), hipMemcpyDeviceToHost);

		timer_stop(&__t);
		fprintf(stderr, "Found %u pairs for two gap enumeration!!\n", countTwoGapEnu);
		fprintf(stderr, "-> Two Gap Enumeration time: %f\n", timer_elapsed(&__t)/1000);      

		//Debugiing!! - Show unsorted onegap pattern
		/*gapPattern2* debugUnsorterPattern2;
		hipHostMalloc((void**)&(debugUnsorterPattern2), countTwoGapEnu*sizeof(gapPattern2));
		hipMemcpy(debugUnsorterPattern2, twogapPattern_d, countTwoGapEnu*sizeof(gapPattern2), hipMemcpyDeviceToHost);
		for(int k=0;k < countTwoGapEnu; k++ ){
		///Debuging see sorted one gap pattern 
		printf("No. %d --> ", k);
		for(int j=0; j< MAX_rule_symbols-4; j++){
		printf("%d ", debugUnsorterPattern2[k].pattern[j]);					
		}
		printf(" |\n");
		///Debugging end				
		}*/
		///
		timer_start(&__t);
		thrust::device_ptr<gappytyp2> twogap_thrust = thrust::device_pointer_cast(twoGap_d);
		thrust::device_ptr<gapPattern2> twogapPattern_thrust = thrust::device_pointer_cast(twogapPattern_d);

		//sort by key! with thrust
		thrust::sort_by_key(twogapPattern_thrust, twogapPattern_thrust+countTwoGapEnu, twogap_thrust, twoGapEnumerationCompare());

		hipMemGetInfo(&freeMem, &totalMem);  		
		fprintf(stderr, "Memory avaliable: Free: %lu, Total: %lu - Two Gap Enu Duplication Memory - After Sort\n",freeMem/(1024*1024), totalMem/(1024*1024));

		//Debugiing!! - Show unsorted onegap pattern		
		/*hipHostMalloc((void**)&(debugUnsorterPattern2), countTwoGapEnu*sizeof(gapPattern2));
		hipMemcpy(debugUnsorterPattern2, twogapPattern_d, countTwoGapEnu*sizeof(gapPattern2), hipMemcpyDeviceToHost);
		for(int k=0;k < countTwoGapEnu; k++ ){
		///Debuging see sorted one gap pattern 
		printf("Sorted No. %d --> ", k);
		for(int j=0; j< MAX_rule_symbols-4; j++){
		printf("ID %d | %d ", debugUnsorterPattern2[k].blockid, debugUnsorterPattern2[k].pattern[j]);					
		}
		printf(" |\n");
		///Debugging end				
		}*/
		///
		
		///Get 1-0 bit array to identify the differences for later sequential scan process
		dim3 grid_two_gap_diff(10, (countTwoGapEnu+10*THREADS_PER_BLOCK_DIFF-1)/(10*THREADS_PER_BLOCK_DIFF));

		uint8_t* zeroOneDiffArrayTwoGap_d;
		hipMalloc((void**)&(zeroOneDiffArrayTwoGap_d), sizeof(uint8_t)*(countTwoGapEnu+1));

		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable after zero one array two gap: Free: %lu, Total: %lu\n",freeMem/(1024*1024), totalMem/(1024*1024));

		zeroOneDiffTwoGap<<<grid_two_gap_diff, block_gap_diff>>>(
				zeroOneDiffArrayTwoGap_d,
				twogapPattern_d,
				countTwoGapEnu);
		timer_stop(&__t);
		fprintf(stderr, "-> Two Gap Enumeration sort time: %f\n", timer_elapsed(&__t)/1000);

		hipDeviceSynchronize();		
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			fprintf(stderr, "CUDA error second After kernel - Zero One Diff - Two gap: %s \n", hipGetErrorString(error));
			exit(-1);
		}	

		uint8_t* zeroOneDiffArrayTwoGap;
		hipHostMalloc((void **)&(zeroOneDiffArrayTwoGap), countTwoGapEnu*sizeof(uint8_t));
		hipMemcpy(zeroOneDiffArrayTwoGap, zeroOneDiffArrayTwoGap_d, countTwoGapEnu*sizeof(uint8_t), hipMemcpyDeviceToHost);

		//////////////////////////////////////
		///Sequential scan! Two Gaps
		////////////////Now get sequential scan done

		//Book keeping
		qryset->twogapcount_enu = countTwoGapEnu;
		hipHostMalloc((void**)&(qryset->twoGap), 
				qryset->twogapcount_enu*sizeof(twogappy));
		hipMemcpy(qryset->twoGap, twoGap_d, 
				qryset->twogapcount_enu*sizeof(twogappy), hipMemcpyDeviceToHost);

		two_gappy_search* twoGapSearch, *twoGapSearch_d;
		hipHostMalloc((void **)&(twoGapSearch), countTwoGapEnu*sizeof(two_gappy_search));// hard coded - can be tune

		//Book keeping
		hipHostMalloc((void**)&(qryset->twogapPattern), 
				qryset->twogapcount_enu*sizeof(twoGapPattern));
		hipMemcpy(qryset->twogapPattern, twogapPattern_d, 
				qryset->twogapcount_enu*sizeof(twoGapPattern), hipMemcpyDeviceToHost);

		timer_start(&__t);
		std::vector<std::vector<unsigned int> > twoGapQueryWithID(qryset->qryscount);
		checkDup.clear();
		//scan
		int distinctTwoGapCount =0;
		initialPos = 0;
		for(int i = 0; i< countTwoGapEnu; i++){
			///Debuging see sorted one gap pattern 
			/*	printf("No. %d --> oneGapId %d %d| number %d ", i, qryset->twogapPattern[i].blockid, 
				qryset->twoGap[i].blockid, qryset->twogapPattern[i].number);
				for(int j=0; j< MAX_rule_symbols-4; j++){
				printf("%d ", qryset->twogapPattern[i].pattern[j]);					
				}
				printf("||| qryend_len %d ||| distinct %d\n", qryset->twoGap[i].qryend_len, distinctTwoGapCount);	*/
			///Debugging end

			if(zeroOneDiffArrayTwoGap[i]==1){
				checkDup.clear();
				initialPos = i;
				twoGapSearch[distinctTwoGapCount].blockid = 
					qryset->twoGap[i].blockid;
				twoGapSearch[distinctTwoGapCount].position = i;
				twoGapSearch[distinctTwoGapCount].qryend_len = qryset->twoGap[i].qryend_len;
				twoGapSearch[distinctTwoGapCount].gap2 = 
					qryset->twoGap[i].gap2;
				twoGapSearch[distinctTwoGapCount].end_on_salist = -1;
				twoGapSearch[distinctTwoGapCount].start_on_salist = -1;
				distinctTwoGapCount ++;					
			} 
			int qid = qryset->tokindex_qryindex[qryset->twoGap[i].gap2];
			if(qid > qryset->qryscount){
				printf("Not possible, Qry count exceeded! in two Gap\n");
				return;
			}
			///remove duplications
			iter_check = checkDup.find(qid);
			if(iter_check == checkDup.end()){
				checkDup.insert(std::make_pair(qid, false));
				twoGapQueryWithID[qid].push_back(distinctTwoGapCount-1); //all to the 
				// uniform id, innitial ID/pos on the two gap enu array
			} 
		}

		/////////Get the sorted stuff out and see gappy phrase		

		timer_stop(&__t);
		fprintf(stderr, "-> Two Gap Enumeration CPU Processing time: %f\n", timer_elapsed(&__t)/1000);      
		fprintf(stderr, "Disticnt two gap pattern: %d\n", distinctTwoGapCount);

		hipFree(twogapPattern_d);
		hipFree(zeroOneDiffArrayTwoGap_d);
		hipFree(twoGap_d);
		hipHostFree(zeroOneDiffArrayTwoGap);

		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable after two gap enumeration finish: Free: %lu, Total: %lu - After One Gap\n",freeMem/(1024*1024), totalMem/(1024*1024));

		//////////Now do the real Suffix Array matching!			
		//////////Now Look For distinct one gappy phrases in Suffix Array
		count = 0;
		hipMemcpy(count_d, &count, sizeof(unsigned int), hipMemcpyHostToDevice);

		hipMalloc((void**)&(twoGapSearch_d), 
				sizeof(two_gappy_search)*distinctTwoGapCount);
		hipMemcpy(twoGapSearch_d, twoGapSearch, 
				sizeof(two_gappy_search)*distinctTwoGapCount, 
				hipMemcpyHostToDevice);

		hipMalloc((void**)&(oneGapSA_d), sizeof(oneGapOnSA)*countOneGapSA);
		hipMemcpy(oneGapSA_d, oneGapSA, sizeof(oneGapOnSA)*countOneGapSA,
				hipMemcpyHostToDevice);

		twoGapOnSA* twoGapSA, *twoGapSA_d;		
		hipMalloc((void**)&(twoGapSA_d), sizeof(twoGapOnSA)*(ONEGAP_PRECOMPUT_PREALLOCATION));

		/////RLP Array memory allocation
		hipMalloc((void**)&(ref_d->RLP), ref_h->toklen*sizeof(unsigned int));
		hipMalloc((void**)&(ref_target_d->L_tar), ref_target_h->toklen*sizeof(uint8_t));
		hipMalloc((void**)&(ref_target_d->R_tar), ref_target_h->toklen*sizeof(uint8_t));
							
		hipMemcpy(ref_target_d->R_tar, ref_target_h->R_tar, ref_target_h->toklen*sizeof(uint8_t), hipMemcpyHostToDevice);
		hipMemcpy(ref_target_d->L_tar, ref_target_h->L_tar, ref_target_h->toklen*sizeof(uint8_t), hipMemcpyHostToDevice);
		hipMemcpy(ref_d->RLP, ref_h->RLP, ref_h->toklen*sizeof(unsigned int), hipMemcpyHostToDevice);
					
		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable right before two gap look up: Free: %lu, Total: %lu - After One Gap\n",freeMem/(1024*1024), totalMem/(1024*1024));

#define THREADS_PER_BLOCK_SA_TWO 128
		dim3 block_gap_twogap_sa(THREADS_PER_BLOCK_SA_TWO, 1);
		dim3 grid_two_gap_sa(500, (distinctTwoGapCount+500-1)/500);

		timer_start(&__t);
		twoGapLookUpSA<<<grid_two_gap_sa, block_gap_twogap_sa>>>(
				ref_d->str,
				ref_d->toklen,
				qryset_d->qrysoffsettok,
				qryset->qryscount,
				qryset->totaltokens,
				qryset_d->result_two,//qryresult
				qryset_d->connectoffset,
				qryset->totalconnect,
				qryset_d->result_connect,
				qryset_d->tokindex_qryindex,
				twoGapSA_d,
				ref_h->precomp_count,
				ref_d->frequentList,
				ref_d->precomp_index,
				ref_d->precomp_onegap,
				count_d,
				twoGapSearch_d,
				distinctTwoGapCount,
				distinctOneGapCount,
				oneGapSearch_d,
				oneGapSA_d,
				countOneGapSA,
				ref_d->RLP,
				ref_target_d->L_tar,
				ref_target_d->R_tar);

		hipDeviceSynchronize();		
		error = hipGetLastError();
		if(error != hipSuccess)
		{
			fprintf(stderr, "CUDA error second After kernel twoGapLookUpSA: %s \n", hipGetErrorString(error));
			exit(-1);
		}	

		unsigned int countTwoGapSA = 0;
		hipMemcpy(&countTwoGapSA, count_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
		fprintf(stderr, "Found %d two gap on SA\n", countTwoGapSA);

		timer_stop(&__t);	
		fprintf(stderr, "-> Two Gap Look Up on SA Kernel: %f\n", timer_elapsed(&__t)/1000);

		hipHostMalloc((void**)&(twoGapSA), countTwoGapSA*sizeof(twoGapOnSA));

		hipFree(twoGapSearch_d);
		hipFree(oneGapSA_d);
		hipFree(oneGapSearch_d);
		hipFree(ref_d->frequentList);
		hipFree(ref_d->precomp_index);	
		hipFree(ref_d->precomp_onegap);
		hipFree(ref_d->RLP);
		hipFree(ref_target_d->L_tar);
		hipFree(ref_target_d->R_tar);
		
		timer_start(&__t);	
		thrust::device_ptr<gappytyp_sa2> twogapSA_thrust = thrust::device_pointer_cast(twoGapSA_d);
		//sort with thrust based on position attribute - ID
		thrust::sort(twogapSA_thrust, twogapSA_thrust+countTwoGapSA, twoGapSACompare());
		
		timer_stop(&__t);
		fprintf(stderr, "-> Two Gap on SA Sorting Kernel: %f\n", timer_elapsed(&__t)/1000);

		hipMemcpy(twoGapSA, twoGapSA_d, countTwoGapSA*sizeof(twoGapOnSA), hipMemcpyDeviceToHost);
		
		timer_start(&__t);
		//Get the start_on_salist/end_on_salist done
		for(int i =0; i<countTwoGapSA; i++ ){
			if (twoGapSA[i].position >= distinctTwoGapCount){
				printf("Counts not POSSIBLE! Wrong sequential scan\n");
				return;
			}
			///Debuging see sorted one gap pattern 
			/*printf("No. %d --> position %d|str_position %d\n", i, twoGapSA[i].position,
					twoGapSA[i].str_position);*/
			///Debugging end
			
			if (twoGapSearch[twoGapSA[i].position].start_on_salist == -1){
				twoGapSearch[twoGapSA[i].position].start_on_salist = i;
			} 

			if (twoGapSearch[twoGapSA[i].position].end_on_salist == -1){
				twoGapSearch[twoGapSA[i].position].end_on_salist = i;
			} else if (twoGapSearch[twoGapSA[i].position].end_on_salist < i){
				twoGapSearch[twoGapSA[i].position].end_on_salist = i;
			}				
		}		
		
		//debugging
		/*for(int i =0; i< distinctTwoGapCount;i++){
			printf("No. %d --> OneGapId %d | SearchTok2 %d | start %d|end %d\n", i, twoGapSearch[i].blockid, twoGapSearch[i].gap2, twoGapSearch[i].start_on_salist,
					twoGapSearch[i].end_on_salist);
			///Debugging end
		}*/
			
		timer_stop(&__t);	
		fprintf(stderr, "-> Two Gap on SA CPU processing time: %f\n", timer_elapsed(&__t)/1000);

		//Book keeping
		hipHostMalloc((void**)&(qryset->twoGapSearch), 
				sizeof(two_gappy_search)*distinctTwoGapCount);
		memcpy(qryset->twoGapSearch, twoGapSearch, sizeof(two_gappy_search)*distinctTwoGapCount);
		qryset->distinctTwoGapCount = distinctTwoGapCount;

		//Book keeping
		hipHostMalloc((void**)&(qryset->twoGapSA), 
				sizeof(twoGapOnSA)*countTwoGapSA);
		memcpy(qryset->twoGapSA, twoGapSA, countTwoGapSA*sizeof(twoGapOnSA));
		qryset->countTwoGapSA = countTwoGapSA;
		qryset->twoGapQueryWithID = twoGapQueryWithID;

		fprintf(stderr, "Suffix Array Kernel Clean Up\n");
		hipFree(twoGapSA_d);			
		hipHostFree(oneGapSearch);
		hipHostFree(oneGapSA);
		hipHostFree(twoGapSA);
		hipHostFree(twoGapSearch);

		hipMemGetInfo(&freeMem, &totalMem);  
		fprintf(stderr, "Memory avaliable: Free: %lu, Total: %lu - End of Suffix.cu\n",freeMem/(1024*1024), totalMem/(1024*1024));
	}

}



////////////////////////////////////////////////	
////Deduplication on SEED one gap precomutation
////////////////////////////////////////////////
/*printf("Start Deduplication - %d\n", qryset->onegapcount);
  timer_start(&__t);
  thrust::device_ptr<gappytyp> dev_ptr = thrust::device_pointer_cast(qryset_d->onegap);

  hipMemGetInfo(&freeMem, &totalMem);  
  fprintf(stderr, "Memory avaliable: Free: %lu, Total: %lu - Duplication Memory - Before Sort\n",freeMem/(1024*1024), totalMem/(1024*1024));

  thrust::sort(dev_ptr, dev_ptr+qryset->onegapcount, oneGapCompare());
  hipMemGetInfo(&freeMem, &totalMem);  

  fprintf(stderr, "Memory avaliable: Free: %lu, Total: %lu - Duplication Memory - After Sort\n",freeMem/(1024*1024), totalMem/(1024*1024));

  hipMemcpy(qryset->onegap, qryset_d->onegap, sizeof(gappy)*qryset->onegapcount, hipMemcpyDeviceToHost);
  hipFree(qryset_d->onegap);*/

//qsort(qryset->onegap, qryset->onegapcount, sizeof(gappy), compareUserTotal_gappy);
//ref_t* ref_hh = &refset->refs[0];

/*for(int i = 0; i< qryset->onegapcount; i++){
  if(qryset->onegap[i].gap+1 > MAX_rule_span){
  printf("WRRRONG! 1 %d\n", i);
  }

  if(qryset->onegap[i].qryend - qryset->onegap[i].qrystart <= 1){
  printf("WRRRONG! 2 %d\n", i);
  }

  if(qryset->onegap[i].qrystart_len + qryset->onegap[i].qryend_len + 1 > MAX_rule_symbols){
  printf("WRRRONG! 3 %d\n", i);
  }	

  if(qryset->tokindex_qryindex[qryset->onegap[i].qrystart] 
  != qryset->tokindex_qryindex[qryset->onegap[i].qryend+qryset->onegap[i].qryend_len-1]){
  printf("WRRRONG! 4 %d\n", i);
  }	

  if(ref_hh->str[qryset->onegap[i].refstr_s] 
  != qryset->qrysoffsettok[qryset->qryscount+qryset->onegap[i].qrystart]
  || ref_hh->str [qryset->onegap[i].refstr_s + qryset->onegap[i].gap] 
  != qryset->qrysoffsettok[qryset->qryscount+qryset->onegap[i].qryend + qryset->onegap[i].qryend_len-1]){
  printf("WRRRONG! 5 %d\n", i);
  }				
  }

  timer_stop(&__t);	
  fprintf(stderr, "-> Deduplication qsort time: %f\n", 
  timer_elapsed(&__t)/1000);*/
////////////////////////////////////////Normal Operations


//thrust::host_vector<gappytyp> h_structures(qryset->onegap, qryset->onegap+qryset->onegapcount);//qryset->onegapcount);//( //(qryset->onegapcount);// = onegap_thrust_h;//(qryset->onegap, qryset->onegap+qryset->onegapcount);// = onegap_thrust_h;
/*
   for(int i = 0; i< qryset->onegapcount; i++){
   h_structures[i].qrystart = qryset->onegap[i].qrystart;
   h_structures[i].qrystart_len = qryset->onegap[i].qrystart_len;
   h_structures[i].qryend =qryset->onegap[i].qryend;
   h_structures[i].qryend_len = qryset->onegap[i].qryend_len;
   h_structures[i].gap = qryset->onegap[i].gap;
   h_structures[i].refstr_s = qryset->onegap[i].refstr_s;
   }*/
//hipFree(raw_ptr);

